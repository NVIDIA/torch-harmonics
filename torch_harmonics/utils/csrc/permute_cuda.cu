#include "hip/hip_runtime.h"
// coding=utf-8
//
// SPDX-FileCopyrightText: Copyright (c) 2025 The torch-harmonics Authors. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
// list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
// this list of conditions and the following disclaimer in the documentation
// and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.


//#include <ATen/cuda/detail/TensorInfo.cuh>
//#include <ATen/cuda/detail/KernelUtils.h>
//#include <ATen/cuda/detail/IndexUtils.cuh>
#include <torch/all.h>

#include <hip/hip_runtime.h>

#include <hipcub/hipcub.hpp>
#include <limits>

#include "cudamacro.h"
#include "permute_cuda.cuh"

// Define the missing macros
#define TRANSP_WARPS_X_TILE_GENERIC (32)
#define TRANSP_WARPS_X_TILE_SM100    (4)

namespace utility_kernels {

    // BEGIN - 4D tensor permutation kernels and functions
__global__ void empty_k() {}

static int getPtxver() {
    hipFuncAttributes attrs;
    CHECK_CUDA(hipFuncGetAttributes(&attrs, reinterpret_cast<const void*>(empty_k)));
    return attrs.ptxVersion*10;
}

torch::Tensor permute_4D_to0231(torch::Tensor src) {

    auto options = torch::TensorOptions().dtype(src.dtype()).device(src.device());
    torch::Tensor dst = torch::empty({src.size(0), src.size(2), src.size(3), src.size(1)}, options);

    const int ptxv = getPtxver();

    // to be further specialized for additional archs, if necessary
    if (ptxv < 100) {
        AT_DISPATCH_FLOATING_TYPES(src.scalar_type(), "permute_to0231_k_tile_generic", ([&] {
            launch_permute_to0231<TRANSP_WARPS_X_TILE_GENERIC, scalar_t>(src, dst);
        }));
        CHECK_ERROR("permute_to0231_k_tile_generic");
    } else {
        AT_DISPATCH_FLOATING_TYPES(src.scalar_type(), "permute_to0231_k_tile_sm100", ([&] {
            launch_permute_to0231<TRANSP_WARPS_X_TILE_SM100, scalar_t>(src, dst);
        }));
        CHECK_ERROR("permute_to0231_k_tile_sm100");
    }

    return dst;
}

torch::Tensor permute_4D_to0312(torch::Tensor src) {

    auto options = torch::TensorOptions().dtype(src.dtype()).device(src.device());
    torch::Tensor dst = torch::empty({src.size(0), src.size(3), src.size(1), src.size(2)}, options);

    const int ptxv = getPtxver();

    // to be further specialized for additional archs, if necessary
    if (ptxv < 100) {
        AT_DISPATCH_FLOATING_TYPES(src.scalar_type(), "permute_to0312_k_tile_generic", ([&] {
            launch_permute_to0312<TRANSP_WARPS_X_TILE_GENERIC, scalar_t>(src, dst);
        }));
        CHECK_ERROR("permute_to0312_k_tile_generic");
    } else {
        AT_DISPATCH_FLOATING_TYPES(src.scalar_type(), "permute_to0312_k_tile_sm100", ([&] {
            launch_permute_to0312<TRANSP_WARPS_X_TILE_SM100, scalar_t>(src, dst);
        }));
        CHECK_ERROR("permute_to0312_k_tile_sm100");
    }

    return dst;
}

TORCH_LIBRARY_IMPL(utility_kernels, CUDA, m)
{
    m.impl("permute_to_0231",  &permute_4D_to0231);
    m.impl("permute_to_0312",  &permute_4D_to0312);
}

// END - tensor permutation kernels and functions

}