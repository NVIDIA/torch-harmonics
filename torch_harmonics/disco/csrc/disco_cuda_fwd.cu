#include "hip/hip_runtime.h"
// coding=utf-8
//
// SPDX-FileCopyrightText: Copyright (c) 2024 The torch-harmonics Authors. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
// list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
// this list of conditions and the following disclaimer in the documentation
// and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "cudamacro.h"
#include "disco.h"
#include "disco_cuda.cuh"
#include "cuda_utils.cuh"

#define THREADS (64)

#define MAX_LOCAL_ARR_LEN (16)

namespace disco_kernels {

using namespace utility_kernels;

void dump_tensor(const char *fname, at::Tensor t);
void dump_csr(const char *fname, at::Tensor roff, at::Tensor cols);
void dump_csr_linear(const char *fname, at::Tensor roff, at::Tensor kers, at::Tensor rows, at::Tensor cols, at::Tensor vals);

template <int BDIM_X, int ELXTH, typename REAL_T>
__device__ void disco_fwd_d(const int Hi, const int Wi, const int K, const int Ho, const int Wo, const int pscale,
                            const int64_t *__restrict__ roff, const int64_t *__restrict__ kers,
                            const int64_t *__restrict__ rows, const int64_t *__restrict__ cols,
                            const REAL_T *__restrict__ vals, const REAL_T *__restrict__ inp, REAL_T *__restrict__ out)
{

    const int tid = threadIdx.x;

    const int64_t bidx = blockIdx.x; // global row
    const int64_t bidy = blockIdx.y; // bc

    int64_t soff = roff[bidx];
    int64_t eoff = roff[bidx + 1];

    const int64_t ker = kers[soff];
    const int64_t row = rows[soff];

    inp += bidy*Hi*Wi;
    out += bidy*K*Ho*Wo + ker*Ho*Wo + row*Wo;

    REAL_T __reg[ELXTH] = {0};

    // align to larger supported fp type
    extern __shared__ __align__(sizeof(double)) unsigned char __sh_ptr[]; // REAL_T __sh[2*Wi + ppscale*(BDIM_X*ELXTH - Wo)]
    REAL_T *__sh = reinterpret_cast<REAL_T *>(__sh_ptr);

    int col_prev = cols[soff];

    int h_prev = col_prev / Wi;
    int w_prev = col_prev % Wi;

    // copy current inp row in shmem
    for (int i = tid; i < Wi; i += BDIM_X) {
        const REAL_T v = inp[h_prev * Wi + i];
        __sh[i] = v;
        __sh[Wi + i] = v;
    }
    // locations __sh[2*Wi : ppscale*(BDIM_X*ELXTH-Wo)] are not used
    __syncthreads();

    // loops along the colums of CTA's row
    for (int64_t nz = soff; nz < eoff; nz++) {

        const int col = cols[nz];
        const REAL_T val = vals[nz];

        // if we are processing a nz with a col value
        // leading to a new row of inp then copy it
        // to shmem;
        // checks whether (h_prev < h) with:
        //  (col >= col_prev - (col_prev % Wi) + Wi)
        if (col >= col_prev - w_prev + Wi) {

            col_prev = col;
            h_prev = col / Wi;
            w_prev = col % Wi;

            __syncthreads();
            for (int i = tid; i < Wi; i += BDIM_X) {
                const REAL_T v = inp[h_prev * Wi + i];
                __sh[i] = v;
                __sh[Wi + i] = v;
            }
            __syncthreads();
        }

        const int w = w_prev + (col - col_prev);

#pragma unroll
        for (int i = 0; i < ELXTH; i++) {

            const int pp = i * BDIM_X + tid;

            // original lines:
            //
            //   if (pp >= Wo) break;
            //   const int wpp = (w + pscale*pp) % Wi;
            //
            // value of (w + pscale*pp) < (Wi + (Wi/Wo)*Wo) = 2*Wi
            // so we can allocate twice the amount of shmem,
            // replicate the current inp row and avoid the costly mod
            //
            // also, to avoid the conditional, sh can be extended to
            // cover the maximum location accessed during this loop
            //
            // REAL_T __sh[2*Wi + ppscale*NUM_REM]
            //
            //   Wi + (Wi/Wo)*BDIM_X*ELXTH = (since BDIM_X*ELXTH >= Wo) =
            // = Wi + (Wi/Wo)*(Wo + (BDIM_X*ELXTH - Wo)) =
            // = 2*Wi + ppscale*NUM_REM
            //
            // with NUM_REM = BDIM_X*ELXTH - Wo

            const int wpp = w + pscale * pp;

            __reg[i] += val * __sh[wpp];
        }
    }

#pragma unroll
    for (int i = 0; i < ELXTH; i++) {

        const int pp = i * BDIM_X + tid;
        if (pp >= Wo) break;

        out[pp] = __reg[i];
    }

    return;
}

template <int BDIM_X, int ELXTH, typename REAL_T>
__global__
    __launch_bounds__(BDIM_X) void disco_fwd_blk_k(const int Hi, const int Wi, const int K, const int Ho, const int Wo,
                                                   const int pscale, const int64_t *__restrict__ roff,
                                                   const int64_t *__restrict__ kers, const int64_t *__restrict__ rows,
                                                   const int64_t *__restrict__ cols, const REAL_T *__restrict__ vals,
                                                   const REAL_T *__restrict__ inp, REAL_T *__restrict__ out)
{

    disco_fwd_d<BDIM_X, ELXTH>(Hi, Wi, K, Ho, Wo, pscale, roff, kers, rows, cols, vals, inp, out);

    return;
}

template <int NTH, int ELXTH, typename REAL_T>
static void launch_kernel(int BC, int Hi, int Wi, int K, int Ho, int Wo, int64_t nrows, int64_t *roff_d, int64_t *ker_d,
                          int64_t *row_d, int64_t *col_d, REAL_T *val_d, REAL_T *inp_d, REAL_T *out_d,
                          hipStream_t stream)
{

    static_assert(sizeof(REAL_T) == 2 || sizeof(REAL_T) == 4 || sizeof(REAL_T) == 8);

    if constexpr (ELXTH <= ELXTH_MAX) {
        if (NTH * ELXTH >= Wo) {
            dim3 grid(nrows, BC);

            const int pscale = Wi / Wo;
            size_t shmem = sizeof(*out_d) * (Wi * 2 + pscale * (NTH * ELXTH - Wo));

            disco_fwd_blk_k<NTH, ELXTH><<<grid, NTH, shmem, stream>>>(Hi, Wi, K, Ho, Wo, pscale, roff_d, ker_d, row_d,
                                                                      col_d, val_d, inp_d, out_d);
        } else {
            launch_kernel<NTH, ELXTH + 1>(BC, Hi, Wi, K, Ho, Wo, nrows, roff_d, ker_d, row_d, col_d, val_d, inp_d,
                                          out_d, stream);
        }
    }
    return;
}


template<typename VAL_T>
__global__ void pack_vals_k(const int64_t K,
                            const int64_t nlat_out,
                            const int64_t *__restrict__ row_off,
                            const VAL_T *__restrict__ val_dat,
                                  VAL_T *__restrict__ val_pck) {

    const int tidx = threadIdx.x;
    const int wid = blockIdx.x*blockDim.y + threadIdx.y;
    if (wid >= nlat_out) {
        return;
    }

    const int64_t rbeg = row_off[wid];
    const int64_t rend = row_off[wid+1];

    const int rlen = rend-rbeg;

    val_pck += rbeg*K;

    for(int off = tidx; off < rlen; off += blockDim.x) {
        for(int ker = 0; ker < K; ker++) {

            val_pck[off*K + ker] = val_dat[ row_off[ker*nlat_out + wid]  + off];
        }
    }

    return;
}
        

// BEGIN VERSION WITH CHANNEL-LAST WITH 2D BLOCKS, 2ND DIM IDENTIFYING CHANNLES, NO EINSUM
#if 1
template<int BDIM_X,
         typename FLOATV_T>
__device__ void processCSR_Kpow2_shm_d(const int wo,
                                       const int rlen,
                                       const int nchan_in, // no. of input floats (not FLOATV_T!) elements along channel dim
                                       const int nlon_in,
                                       const int pscale,
                                       const int K,
                                       const float    *__restrict__ x,
                                       const int64_t  *__restrict__ cols,
                                       const FLOATV_T *__restrict__ vals,
                                             FLOATV_T *__restrict__ shy) {
    const int tidx = threadIdx.x;

    // only used in K_POWER_2==1 branch
    const int log2_K = __ffs(K)-1;

    x    += tidx >> log2_K;
    vals += tidx & (K-1);

    const int BDIM_XdivK = BDIM_X >> log2_K;

    for(int off = 0; off < rlen; off++) {

        const int64_t  col = cols[off];

        const int hi = col / nlon_in;
        const int wi = col - (hi*nlon_in);

        //const int wip = (wi + pscale*wo) % nlon_in;
        // value of (wi + pscale*wo) < (Wi + (Wi/Wo)*Wo) = 2*Wi
        // so we can replace the modulo with:
        int wip = wi + pscale*wo;
        if (wip >= nlon_in) wip -= nlon_in;

        const float *_x = x + int64_t(hi)*nlon_in*nchan_in + int64_t(wip)*nchan_in;

        // if BDIM_X is a multiple of K then "i*(j*BDIM_X) % K = const",
        // so thread "i" only needs to read vals[off*K + (i % K)] to update the
        // whole channel array
    
        const FLOATV_T myval = vals[0]; //vals[off*K + tidxModK];
        
        for(int chan = tidx; chan < nchan_in*K; chan += BDIM_X) { // no. of vectors in nchan_in*K dim on intermediate out

            shy[chan] = __vadd(shy[chan],
                               __vmul(myval,
                                      __vset<FLOATV_T>(_x[0])));
            _x += BDIM_XdivK;
        }

        vals += K;
    }
    return;
}

template<int BDIM_X,
         typename FLOATV_T>
__device__ void processCSR_Kanyv_shm_d(const int wo,
                                       const int rlen,
                                       const int nchan_in, // no. of input floats (not FLOATV_T!) elements along channel dim
                                       const int nlon_in,
                                       const int pscale,
                                       const int K,
                                       const float    *__restrict__ x,
                                       const int64_t  *__restrict__ cols,
                                       const FLOATV_T *__restrict__ vals,
                                             FLOATV_T *__restrict__ shy) {
    const int tidx = threadIdx.x;

    for(int off = 0; off < rlen; off++) {

        const int64_t  col = cols[off];

        const int hi = col / nlon_in;
        const int wi = col - (hi*nlon_in);

        //const int wip = (wi + pscale*wo) % nlon_in;
        // value of (wi + pscale*wo) < (Wi + (Wi/Wo)*Wo) = 2*Wi
        // so we can replace the modulo with:
        int wip = wi + pscale*wo;
        if (wip >= nlon_in) wip -= nlon_in;

        const float *_x = x + int64_t(hi)*nlon_in*nchan_in + int64_t(wip)*nchan_in;

        // if BDIM_X is not a multiple of K then "i*(j*BDIM_X) % K = f(i,j)",
        // so the mod need to be recomputed at each iteration of update the update loop
        for(int chan = tidx; chan < nchan_in*K; chan += BDIM_X) { // no. of vectors in nchan_in*K dim on intermediate out

            const int iDivK = chan / K;
            const int iModK = chan - (iDivK*K);
        
            shy[chan] = __vadd(shy[chan],
                               __vmul(vals[iModK],
                                      __vset<FLOATV_T>(_x[iDivK])));
        }
        
        vals += K;
    }
    return;
}

template<int BDIM, // change to BDIM_X(<=WARP_SIZE), BDIM_Y TO HANDLE SMALL CHXGR_IN
         typename FLOATV_T> // either float or float4
__global__
__launch_bounds__(BDIM)
void s2_disco_fwd_generic_vec_k(int nchan_in,   // no. of input  float (not FLOATV_T!) elements along channel dim
                                int nlat_in,
                                int nlon_in,
                                int nlat_out,
                                int nlon_out,
                                int pscale,
                                int K,          // no. of output FLOATV_T elem along K dim (kernel size)
                                const float   *__restrict__ x,
                                const int32_t *__restrict__ row_idx,
                                const int64_t *__restrict__ row_off,
                                const int64_t *__restrict__ col_idx,
                                const FLOATV_T *__restrict__ val_pck,
                                      FLOATV_T *__restrict__ y) {

    constexpr int VEC_SIZE = sizeof(FLOATV_T) / sizeof(float);

    const int tidx  = threadIdx.x;

    const int batch = blockIdx.y;
    const int ctaid = blockIdx.x*blockDim.y + threadIdx.y;

    if (ctaid >= nlat_out*nlon_out) {
        return;
    }

#if 1
    const int h = ctaid / nlon_out;
    const int wo = ctaid - (h*nlon_out);
    const int ho = row_idx[h];
#else
    // for now don't use row_idx
    const int ho = wid / nlon_out;
    const int wo = wid - (ho*nlon_out);
#endif
    
    const int nchan_out = nchan_in*K;

    extern __shared__ __align__(sizeof(float4)) float shext[];
    FLOATV_T *shy = reinterpret_cast<FLOATV_T *>(shext) + threadIdx.y*nchan_out;
    
    for(int chan = tidx; chan < nchan_out; chan += WARP_SIZE) {
        shy[chan] = __vset<FLOATV_T>(0.f);
    }

    x += int64_t(batch)*nlat_in*nlon_in*nchan_in;
    y += int64_t(batch)*nlat_out*nlon_out*nchan_out + int64_t(ho)*nlon_out*nchan_out + int64_t(wo)*nchan_out;

    const int64_t rbeg = row_off[ho];
    const int64_t rend = row_off[ho+1];

    col_idx += rbeg;
    val_pck += rbeg*K; // val_pck CSR contains K values per element

    const int rlen = rend-rbeg;

    // check if BDIM_X is a multiple of K; since BDIM_X is a power of 2, check if K is also a power of two
    if (!(K & K-1)) { processCSR_Kpow2_shm_d<WARP_SIZE>(wo, rlen, nchan_in, nlon_in, pscale, K, x, col_idx, val_pck, shy); }
    else            { processCSR_Kanyv_shm_d<WARP_SIZE>(wo, rlen, nchan_in, nlon_in, pscale, K, x, col_idx, val_pck, shy); }

    for(int chan = tidx; chan < nchan_out; chan += WARP_SIZE) {
        y[chan] = shy[chan];
    }

    return;
}

template<int BDIM_X,
         int SHPAD,
         int NLOC,
         typename FLOATV_T>
__device__ void processCSR_Kpow2_reg_d(const int wo,
                                       const int rlen,
                                       const int nchan_in,    // no. of input floats (not FLOATV_T!) elements along channel dim
                                       const int nlon_in,
                                       const int pscale,
                                       const int K,           // kernel size
                                       const float    *__restrict__ x,
                                       const int64_t  *__restrict__ cols,
                                       const FLOATV_T *__restrict__ vals,
                                       const float *(&shXOff)[BDIM_X+SHPAD],
                                             FLOATV_T (&locy)[NLOC]) {

    constexpr int NLOC_M1 = NLOC-1;

    const int tidx = threadIdx.x;
    const int tidy = threadIdx.y;

    // unused if BDIM_X > WARP_SIZE
    unsigned int subwarp_mask = FULL_MASK;

    if constexpr(BDIM_X <= WARP_SIZE) {
        constexpr unsigned int MASK = (1ull << BDIM_X)-1;
        subwarp_mask = MASK << (tidy*BDIM_X);
    }

    // only used in K_POWER_2==1 branch
    const int log2_K = __ffs(K)-1;

    const int tidxDivK = tidx >> log2_K;
    const int tidxModK = tidx  & (K-1);
    
    cols += tidx;
    vals += tidxModK;

    const int BDIM_XdivK = BDIM_X >> log2_K;

    for(int off = 0; off < rlen; off++) {

        if ((off % BDIM_X) == 0) {
            if constexpr(BDIM_X <= WARP_SIZE) { __syncwarp(subwarp_mask); }
            else                              { __syncthreads(); }

            const int64_t  col = (off+tidx < rlen) ? cols[0] : 0;

            const int hi = col / nlon_in;
            const int wi = col - (hi*nlon_in);

            //const int wip = (wi + pscale*wo) % nlon_in;
            // value of (wi + pscale*wo) < (Wi + (Wi/Wo)*Wo) = 2*Wi
            // so we can replace the modulo with:
            int wip = wi + pscale*wo;
            if (wip >= nlon_in) wip -= nlon_in;

            shXOff[tidx] = x + int64_t(hi)*nlon_in*nchan_in + int64_t(wip)*nchan_in;
            cols += BDIM_X;

            if constexpr(BDIM_X <= WARP_SIZE) { __syncwarp(subwarp_mask); }
            else                              { __syncthreads(); }
        }

        const float *_x = shXOff[off % BDIM_X] + tidxDivK;

        // if BDIM_X is a multiple of K then "i*(j*BDIM_X) % K = const",
        // so thread "i" only needs to read vals[off*K + (i % K)] to update the
        // whole channel array
    
        #pragma unroll
        for(int i = 0; i < NLOC_M1; i++) {
            locy[i] = __vadd(locy[i],
                             __vmul(vals[0],
                                    __vset<FLOATV_T>(_x[i*BDIM_XdivK])));
        }
        if (NLOC_M1*BDIM_X+tidx < nchan_in*K) {
            locy[NLOC_M1] = __vadd(locy[NLOC_M1],
                                   __vmul(vals[0],
                                          __vset<FLOATV_T>(_x[NLOC_M1*BDIM_XdivK])));
        }

        vals += K;
    }
    return;
}

template<int BDIM_X,
         int SHPAD,
         int NLOC,
         typename FLOATV_T>
__device__ void processCSR_Kanyv_reg_d(const int wo,
                                       const int rlen,
                                       const int nchan_in,    // no. of input floats (not FLOATV_T!) elements along channel dim
                                       const int nlon_in,
                                       const int pscale,
                                       const int K,           // kernel size
                                       const float    *__restrict__ x,
                                       const int64_t  *__restrict__ cols,
                                       const FLOATV_T *__restrict__ vals,
                                       const float *(&shXOff)[BDIM_X+SHPAD],
                                             FLOATV_T (&locy)[NLOC]) {

    constexpr int NLOC_M1 = NLOC-1;

    const int tidx = threadIdx.x;
    const int tidy = threadIdx.y;

    // unused if BDIM_X > WARP_SIZE
    unsigned int subwarp_mask = 0xFFFFFFFF;

    if constexpr(BDIM_X <= WARP_SIZE) {
        constexpr unsigned int MASK = (1ull << BDIM_X)-1;
        subwarp_mask = MASK << (tidy*BDIM_X);
    }

    cols += tidx;

    for(int off = 0; off < rlen; off++) {

        if ((off % BDIM_X) == 0) {
            if constexpr(BDIM_X <= WARP_SIZE) { __syncwarp(subwarp_mask); }
            else                              { __syncthreads(); }

            const int64_t  col = (off+tidx < rlen) ? cols[0] : 0;

            const int hi = col / nlon_in;
            const int wi = col - (hi*nlon_in);

            //const int wip = (wi + pscale*wo) % nlon_in;
            // value of (wi + pscale*wo) < (Wi + (Wi/Wo)*Wo) = 2*Wi
            // so we can replace the modulo with:
            int wip = wi + pscale*wo;
            if (wip >= nlon_in) wip -= nlon_in;

            shXOff[tidx] = x + int64_t(hi)*nlon_in*nchan_in + int64_t(wip)*nchan_in;
            cols += BDIM_X;

            if constexpr(BDIM_X <= WARP_SIZE) { __syncwarp(subwarp_mask); }
            else                              { __syncthreads(); }
        }

        const float *_x = shXOff[off % BDIM_X];

        // if BDIM_X is not a multiple of K then "i*(j*BDIM_X) % K = f(i,j)",
        // so the mod need to be recomputed at each iteration of update the update loop
        
        #pragma unroll
        for(int i = 0; i < NLOC_M1; i++) {

            const int chan = i*BDIM_X+tidx;
            const int iDivK = chan / K;
            const int iModK = chan - (iDivK*K);

            const FLOATV_T vval = vals[iModK]; //vals[off*K + iModK];
            const FLOATV_T xval = __vset<FLOATV_T>(_x[iDivK]);

            locy[i] = __vadd(locy[i], __vmul(vval, xval));
        }
        if (NLOC_M1*BDIM_X+tidx < nchan_in*K) {
            
            const int chan = NLOC_M1*BDIM_X+tidx;
            const int iDivK = chan / K;
            const int iModK = chan - (iDivK*K);

            const FLOATV_T vval = vals[iModK]; //vals[off*K + iModK];
            const FLOATV_T xval = __vset<FLOATV_T>(_x[iDivK]);

            locy[NLOC_M1] = __vadd(locy[NLOC_M1], __vmul(vval, xval));
        }
        
        vals += K;
    }
    return;
}

template<int BDIM_X,
         int BDIM_Y,
         int NLOC,
         typename FLOATV_T> // either float or float4
__global__
__launch_bounds__(BDIM_X*BDIM_Y)
void s2_disco_fwd_special_vec_k(const int nchan_in,   // no. of input  float (not FLOATV_T!) elements along channel dim
                                const int nlat_in,
                                const int nlon_in,
                                const int nlat_out,
                                const int nlon_out,
                                const int pscale,
                                const int K,          // no. of output FLOATV_T elem along K dim (kernel size)
                                const float   *__restrict__ x,
                                const int32_t *__restrict__ row_idx,
                                const int64_t *__restrict__ row_off,
                                const int64_t *__restrict__ col_idx,
                                const FLOATV_T *__restrict__ val_pck,
                                      FLOATV_T *__restrict__ y) {

    static_assert(0 == (BDIM_X & (BDIM_X-1)));
    static_assert(0 == (BDIM_Y & (BDIM_Y-1)));
    static_assert((BDIM_X <= 32 && BDIM_Y  > 1) ||
                  (BDIM_X  > 32 && BDIM_Y == 1)) ;

    constexpr int NLOC_M1 = NLOC-1;

    constexpr int VEC_SIZE = sizeof(FLOATV_T) / sizeof(float);

    const int tidx  = threadIdx.x;
    const int tidy  = threadIdx.y;

    const int batch = blockIdx.y;
    const int ctaid = blockIdx.x*blockDim.y + threadIdx.y;

    if (ctaid >= nlat_out*nlon_out) {
        return;
    }

#if 1
    const int h = ctaid / nlon_out;
    const int wo = ctaid - (h*nlon_out);
    const int ho = row_idx[h];
#else
    // for now don't use row_idx
    const int ho = ctaid / nlon_out;
    const int wo = ctaid - (ho*nlon_out);
#endif

    const int nchan_out = nchan_in*K;

    FLOATV_T locy[NLOC];

    x += int64_t(batch)*nlat_in*nlon_in*nchan_in;
    y += int64_t(batch)*nlat_out*nlon_out*nchan_out + int64_t(ho)*nlon_out*nchan_out + int64_t(wo)*nchan_out + tidx;

    #pragma unroll
    for(int i = 0; i < NLOC; i++) {
        locy[i] = __vset<FLOATV_T>(0.f);
    }

    const int64_t rbeg = row_off[ho];
    const int64_t rend = row_off[ho+1];

    col_idx += rbeg;
    val_pck += rbeg*K; // val_pck CSR contains K values per element

    const int rlen = rend-rbeg;

    constexpr int PAD = (BDIM_X < WARP_SIZE) ? 1 : 0;
    __shared__ const float *shXOffAll[BDIM_Y][BDIM_X+PAD];

    // check if BDIM_X is a multiple of K; since BDIM_X is a power of 2, check if K is also a power of two
    const int isKpow2 = !(K & (K-1));
    if (isKpow2) { processCSR_Kpow2_reg_d<BDIM_X, PAD, NLOC>(wo, rlen, nchan_in, nlon_in, pscale, K, x, col_idx, val_pck, shXOffAll[tidy], locy); }
    else         { processCSR_Kanyv_reg_d<BDIM_X, PAD, NLOC>(wo, rlen, nchan_in, nlon_in, pscale, K, x, col_idx, val_pck, shXOffAll[tidy], locy); }


    #pragma unroll
    for(int i = 0; i < NLOC_M1; i++) {
        y[i*BDIM_X] = locy[i];
    }
    if (NLOC_M1*BDIM_X+tidx < nchan_out) {
        y[NLOC_M1*BDIM_X] = locy[NLOC_M1];
    }

    return;
}

template<typename FLOATV_T>
void launch_gen_disco_fwd(int64_t batch_size,
                          int64_t nchan_in,
                          int64_t nlat_in,
                          int64_t nlon_in,
                          int64_t nlat_out,
                          int64_t nlon_out,
                          int64_t K,
                          float *__restrict__ _xp,
                          int32_t *_row_idx,
                          int64_t *_row_off,
                          int64_t *_col_idx,
                          FLOATV_T *_val_pck,
                          FLOATV_T *__restrict__ _yp,
                          hipStream_t stream) {

    dim3 block(WARP_SIZE, THREADS/WARP_SIZE);
    dim3 grid(DIV_UP(nlat_out*nlon_out, block.y), batch_size);

    size_t shsize = sizeof(FLOATV_T)*(nchan_in*K)*block.y;

    const int pscale = nlon_in / nlon_out;
#if 0
    printf("Launching s2_disco_fwd_generic_vec_k<%d, float%s><<<..., ..., %zu, ...>>> with:\n"
           "\tngroup: %ld\n"
           "\tnchan_in: %ld\n"
           "\tK: %ld\n\n",
           THREADS, sizeof(FLOATV_T)==16?"4":"", shsize, ngroup, nchan_in, K);
#endif
    // will use only the first 1/K-th of the CSR, i.e. only the first nlat_out rows
    s2_disco_fwd_generic_vec_k<THREADS>
                              <<<grid, block, shsize, stream>>>(nchan_in, nlat_in, nlon_in, nlat_out, nlon_out, pscale, K,
                                                                _xp, _row_idx, _row_off, _col_idx, _val_pck, _yp);
    CHECK_ERROR("s2_disco_fwd_generic_vec_k");

    return;
}

template<int BDIM_X,
         int CUR_LOC_SIZE,
         int MAX_LOC_SIZE, // max size of FLOATV_T[] local array
         typename FLOATV_T>
void launch_spc_disco_fwd(int nloc,      // "BDIM_X*nloc" >= nchans
                          int64_t batch_size,
                          int64_t nchan_in,
                          int64_t nlat_in,
                          int64_t nlon_in,
                          int64_t nlat_out,
                          int64_t nlon_out,
                          int64_t K,
                          float *__restrict__ _xp,
                          int32_t *_row_idx,
                          int64_t *_row_off,
                          int64_t *_col_idx,
                          FLOATV_T *_val_pck,
                          FLOATV_T *__restrict__ _yp,
                          hipStream_t stream) {

    if (CUR_LOC_SIZE == nloc) {

        // block size set to 64 threads
        constexpr int BDIM_Y = (BDIM_X <= WARP_SIZE) ? THREADS / BDIM_X : 1;

        // groups in gridDim.y
        dim3 block(BDIM_X, BDIM_Y);
        dim3 grid(DIV_UP(nlat_out*nlon_out, block.y), batch_size);
    
        size_t shsize = 0; //sizeof(float)*chxgrp_out * block.y;

        const int pscale = nlon_in / nlon_out;
#if 0
        printf("Launching s2_disco_fwd_special_vec_k<%d, %d, %d, float%s><<<(%d, %d, %d), (%d, %d), ..., %zu, ...>>> with:\n"
               "\tngroup: %ld\n"
               "\tnchan_in: %ld\n"
               "\tK: %ld\n\n",
               BDIM_X, BDIM_Y, CUR_LOC_SIZE, sizeof(FLOATV_T)==16?"4":"", grid.x, grid.y, grid.z, block.x, block.y, shsize, ngroup, nchan_in, K);
#endif
        s2_disco_fwd_special_vec_k<BDIM_X, BDIM_Y, CUR_LOC_SIZE>
                                  <<<grid, block, shsize, stream>>>(nchan_in, nlat_in, nlon_in, nlat_out, nlon_out, pscale, K,
                                                                    _xp, _row_idx, _row_off, _col_idx, _val_pck, _yp);

        CHECK_ERROR("s2_disco_fwd_special_vec_k");

        return;
    }
    if constexpr(CUR_LOC_SIZE < MAX_LOC_SIZE) {
         launch_spc_disco_fwd<BDIM_X,
                              CUR_LOC_SIZE+1,
                              MAX_LOC_SIZE>(nloc, batch_size, nchan_in, nlat_in, nlon_in, nlat_out, nlon_out,
                                            K, _xp, _row_idx, _row_off, _col_idx, _val_pck, _yp, stream);
    }
    return;
}

static void s2_disco_fwd_dispatch(int64_t batch_size,
                                  int64_t nchan_in,
                                  int64_t nlon_in,
                                  int64_t nlat_out,
                                  int64_t nlon_out,
                                  int64_t K,
                                  at::Tensor xP,
                                  at::Tensor row_off, // CSR row offsets
                                  at::Tensor col_idx, // CSR column indices
                                  at::Tensor val_dat, // CSR value data
                                  at::Tensor yP) {

    static_assert(0 == (MAX_LOCAL_ARR_LEN & (MAX_LOCAL_ARR_LEN-1)));

    if (batch_size <= 0 ||
        nchan_in   <= 0 ||
        nlon_in    <= 0 ||
        nlat_out   <= 0 ||
        nlon_out   <= 0 ||
        K          <= 0) {

            fprintf(stderr,
                    ":%s:%d: invalid value of one or more input parameters!\n", 
                    __FILE__, __LINE__);
            exit(EXIT_FAILURE);
    }

    // get stream
    auto stream = at::cuda::getCurrentCUDAStream().stream();

    // sort row indices (ho-s) in descending order
    // based on (row_off[ho+1]-row_off[ho])
    at::Tensor row_idx = sortRows(nlat_out, row_off, stream);


    // replace the K sequential CRSs in "val_dat":
    //
    //  val_dat[    0:  nnz/K) for ker = 0
    //  val_dat[nnz/K:2*nnz/K) for ker = 1
    //  ...
    //  val_dat[nnz/K:2*nnz/K) for ker = K-1
    //
    // with a packed CSR:
    // 
    //  val_dat[nnz/K][K], i.e. with a CSR where elements of the original K CSRs are packed in consecutive elements
    assert(0 == (val_idx.size(0) % K));

    // move into "disco_cuda_utils.cu" IF val_dat format won't be changed upstream in the call chain
    int64_t val_dims[] = {val_dat.size(0)};
    auto options = torch::TensorOptions().device(val_dat.device()).dtype(val_dat.dtype());
    torch::Tensor val_pck = torch::zeros(val_dims, options);
    {
        dim3 block(WARP_SIZE, THREADS/WARP_SIZE);
        dim3 grid(DIV_UP(nlat_out, block.y));

        pack_vals_k<<<grid, block, 0, stream>>>(K, nlat_out,
                                                row_off.data_ptr<int64_t>(),
                                                val_dat.data_ptr<float>(),
                                                val_pck.data_ptr<float>());
    }
    // if K is a multiple of VEC_SIZE it will be read with vector lds

    const int nlat_in = xP.size(1);

    // smallest power of two "bdimx" (>=4) s.t. bdimx*MAX_LOCAL_ARR_LEN >= nchan_in*K
    int bdimx;
    bdimx = DIV_UP(nchan_in*K, MAX_LOCAL_ARR_LEN);
    bdimx = max(bdimx, WARP_SIZE/8); // min 4 threads per group
    bdimx = next_pow2(bdimx);

    float *_xp = reinterpret_cast<float *>(xP.data_ptr());
    float *_yp = reinterpret_cast<float *>(yP.data_ptr());

    int32_t *_row_idx = reinterpret_cast<int32_t *>(row_idx.data_ptr());
    int64_t *_row_off = reinterpret_cast<int64_t *>(row_off.data_ptr());
    int64_t *_col_idx = reinterpret_cast<int64_t *>(col_idx.data_ptr());
    float   *_val_pck = reinterpret_cast<float   *>(val_pck.data_ptr());

    constexpr int VEC_SIZE = sizeof(float4) / sizeof(float);

    if (!is_aligned<sizeof(float4)>(_yp)      ||
        !is_aligned<sizeof(float4)>(_val_pck) ||
        (K % VEC_SIZE) != 0) {

        //printf("%s:%d: VEC_SIZE: %d, nchan_in: %d, K: %d, _xp: %p, _yp: %p\n", __func__, __LINE__, VEC_SIZE, nchan_in, K, _xp, _yp);

        const int nloc = DIV_UP(nchan_in*K, bdimx);
        
        // to avoid the compilation of unused template instances;
        // we use a block size BDIM_X that is the smallest power of 2
        // such that BDIM_X*MAX_LOCAL_ARR_LEN >= nchan_in*K, so
        // BDIM_X > 32 are used only for:
        //
        //  (BDIM_X-1)*MAX_LOCAL_ARR_LEN < nchan <= BDIM_X*MAX_LOCAL_ARR_LEN
        constexpr int MIN_LOC_ARR_LEN = MAX_LOCAL_ARR_LEN/2+1;

        // use 2D blocks only if 32 threads are enough
        switch(bdimx) {
            case    8: launch_spc_disco_fwd<   8,               1, MAX_LOCAL_ARR_LEN>(nloc, batch_size, nchan_in, nlat_in, nlon_in, nlat_out, nlon_out, K, _xp, _row_idx, _row_off, _col_idx, _val_pck, _yp, stream); break;
            case   16: launch_spc_disco_fwd<  16, MIN_LOC_ARR_LEN, MAX_LOCAL_ARR_LEN>(nloc, batch_size, nchan_in, nlat_in, nlon_in, nlat_out, nlon_out, K, _xp, _row_idx, _row_off, _col_idx, _val_pck, _yp, stream); break;
            case   32: launch_spc_disco_fwd<  32, MIN_LOC_ARR_LEN, MAX_LOCAL_ARR_LEN>(nloc, batch_size, nchan_in, nlat_in, nlon_in, nlat_out, nlon_out, K, _xp, _row_idx, _row_off, _col_idx, _val_pck, _yp, stream); break;
            case   64: launch_spc_disco_fwd<  64, MIN_LOC_ARR_LEN, MAX_LOCAL_ARR_LEN>(nloc, batch_size, nchan_in, nlat_in, nlon_in, nlat_out, nlon_out, K, _xp, _row_idx, _row_off, _col_idx, _val_pck, _yp, stream); break;
            case  128: launch_spc_disco_fwd< 128, MIN_LOC_ARR_LEN, MAX_LOCAL_ARR_LEN>(nloc, batch_size, nchan_in, nlat_in, nlon_in, nlat_out, nlon_out, K, _xp, _row_idx, _row_off, _col_idx, _val_pck, _yp, stream); break;
            case  256: launch_spc_disco_fwd< 256, MIN_LOC_ARR_LEN, MAX_LOCAL_ARR_LEN>(nloc, batch_size, nchan_in, nlat_in, nlon_in, nlat_out, nlon_out, K, _xp, _row_idx, _row_off, _col_idx, _val_pck, _yp, stream); break;
            case  512: launch_spc_disco_fwd< 512, MIN_LOC_ARR_LEN, MAX_LOCAL_ARR_LEN>(nloc, batch_size, nchan_in, nlat_in, nlon_in, nlat_out, nlon_out, K, _xp, _row_idx, _row_off, _col_idx, _val_pck, _yp, stream); break;
            case 1024: launch_spc_disco_fwd<1024, MIN_LOC_ARR_LEN, MAX_LOCAL_ARR_LEN>(nloc, batch_size, nchan_in, nlat_in, nlon_in, nlat_out, nlon_out, K, _xp, _row_idx, _row_off, _col_idx, _val_pck, _yp, stream); break;
            default:   launch_gen_disco_fwd                                          (      batch_size, nchan_in, nlat_in, nlon_in, nlat_out, nlon_out, K, _xp, _row_idx, _row_off, _col_idx, _val_pck, _yp, stream); break;
        }

    } else {

        //printf("%s:%d: VEC_SIZE: %d, nchan_in: %d, K: %d, _xp: %p, _yp: %p\n", __func__, __LINE__, VEC_SIZE, nchan_in, K, _xp, _yp);

        //float4 *_xp4 = reinterpret_cast<float4 *>(_xp);
        float4 *_yp4 = reinterpret_cast<float4 *>(_yp);

        float4 *_val_pck4 = reinterpret_cast<float4 *>(_val_pck);

        K /= VEC_SIZE;
        const int nloc = DIV_UP(nchan_in*K, bdimx);

        constexpr int MAX_LOCAL_VEC_LEN = MAX_LOCAL_ARR_LEN / VEC_SIZE;
        constexpr int MIN_LOC_VEC_LEN = MAX_LOCAL_VEC_LEN/2+1;

        // use 2D blocks only if 32 threads are enough
        switch(bdimx) {
            case    8: launch_spc_disco_fwd<   8,               1, MAX_LOCAL_VEC_LEN>(nloc, batch_size, nchan_in, nlat_in, nlon_in, nlat_out, nlon_out, K, _xp, _row_idx, _row_off, _col_idx, _val_pck4, _yp4, stream); break;
            case   16: launch_spc_disco_fwd<  16, MIN_LOC_VEC_LEN, MAX_LOCAL_VEC_LEN>(nloc, batch_size, nchan_in, nlat_in, nlon_in, nlat_out, nlon_out, K, _xp, _row_idx, _row_off, _col_idx, _val_pck4, _yp4, stream); break;
            case   32: launch_spc_disco_fwd<  32, MIN_LOC_VEC_LEN, MAX_LOCAL_VEC_LEN>(nloc, batch_size, nchan_in, nlat_in, nlon_in, nlat_out, nlon_out, K, _xp, _row_idx, _row_off, _col_idx, _val_pck4, _yp4, stream); break;
            case   64: launch_spc_disco_fwd<  64, MIN_LOC_VEC_LEN, MAX_LOCAL_VEC_LEN>(nloc, batch_size, nchan_in, nlat_in, nlon_in, nlat_out, nlon_out, K, _xp, _row_idx, _row_off, _col_idx, _val_pck4, _yp4, stream); break;
            case  128: launch_spc_disco_fwd< 128, MIN_LOC_VEC_LEN, MAX_LOCAL_VEC_LEN>(nloc, batch_size, nchan_in, nlat_in, nlon_in, nlat_out, nlon_out, K, _xp, _row_idx, _row_off, _col_idx, _val_pck4, _yp4, stream); break;
            case  256: launch_spc_disco_fwd< 256, MIN_LOC_VEC_LEN, MAX_LOCAL_VEC_LEN>(nloc, batch_size, nchan_in, nlat_in, nlon_in, nlat_out, nlon_out, K, _xp, _row_idx, _row_off, _col_idx, _val_pck4, _yp4, stream); break;
            case  512: launch_spc_disco_fwd< 512, MIN_LOC_VEC_LEN, MAX_LOCAL_VEC_LEN>(nloc, batch_size, nchan_in, nlat_in, nlon_in, nlat_out, nlon_out, K, _xp, _row_idx, _row_off, _col_idx, _val_pck4, _yp4, stream); break;
            case 1024: launch_spc_disco_fwd<1024, MIN_LOC_VEC_LEN, MAX_LOCAL_VEC_LEN>(nloc, batch_size, nchan_in, nlat_in, nlon_in, nlat_out, nlon_out, K, _xp, _row_idx, _row_off, _col_idx, _val_pck4, _yp4, stream); break;
            default:   launch_gen_disco_fwd                                          (      batch_size, nchan_in, nlat_in, nlon_in, nlat_out, nlon_out, K, _xp, _row_idx, _row_off, _col_idx, _val_pck4, _yp4, stream); break;
        }
    }
    return;
}
#endif
// END VERSION WITH CHANNEL-LAST WITH 2D BLOCKS, 2ND DIM IDENTIFYING CHANNLES, NO EINSUM








    // utility functions
    void dump_out_kers(const char *fprefix, at::Tensor t) {

        int64_t B = t.size(0);
        int64_t C = t.size(1);
        int64_t K = t.size(2);
        int64_t Ho = t.size(3);
        int64_t Wo = t.size(4);

        at::Tensor t_h = t.to(torch::kCPU);

        auto accessor = t_h.accessor<float, 5>();

        printf("Writing data to file...");

        char fname[256];

        for(size_t k = 0; k < K; k++) {
            
            snprintf(fname, sizeof(fname), "%s_%ld.txt", fprefix, k);

            FILE *fp = fopen(fname, "w");
            if (!fp) {
                    fprintf(stderr, "Cannot open file %s for writing!\n", fname);
                    exit(EXIT_FAILURE);
            }
            for(int64_t b = 0; b < B; b++) {
                fprintf(fp, "b: %ld\n", b);
                for(int64_t c = 0; c < C; c++) {
                    fprintf(fp, "c: %ld\n", c);
                    for(int64_t h = 0; h < Ho; h++) {
                        for(int64_t w = 0; w < Wo; w++) {
                            fprintf(fp, " %f", accessor[b][c][k][h][w]);
                        }
                        fprintf(fp, "\n");
                    }
                    fprintf(fp, "\n");
                }
                fprintf(fp, "\n");
            }
            fclose(fp);
        }
        printf("done\n");

        return;
    }

    torch::Tensor disco_cuda_fwd(torch::Tensor inp, torch::Tensor roff_idx, torch::Tensor ker_idx, torch::Tensor row_idx,
                                 torch::Tensor col_idx, torch::Tensor val, int64_t K, int64_t Ho, int64_t Wo)
    {

        // some sanity checks
        CHECK_CUDA_INPUT_TENSOR(inp);
        CHECK_CUDA_INPUT_TENSOR(roff_idx);
        CHECK_CUDA_INPUT_TENSOR(ker_idx);
        CHECK_CUDA_INPUT_TENSOR(row_idx);
        CHECK_CUDA_INPUT_TENSOR(col_idx);
        CHECK_CUDA_INPUT_TENSOR(val);

        // assume input is B, H, W, C
        int64_t B = inp.size(0);
        int64_t Hi = inp.size(1);
        int64_t Wi = inp.size(2);
        int64_t C = inp.size(3);
        //int64_t BC = B * C;
        int64_t nrows = roff_idx.size(0) - 1;

        // rename dimensions consistent with attention
        int64_t batch_size = B;
        int64_t nchan = C;
        int64_t nlat_in = Hi;
        int64_t nlon_in = Wi;
        int64_t nlat_out = Ho;
        int64_t nlon_out = Wo;
/*
        int64_t ngroup = 1;
        if (std::getenv("S2_NGROUP")) {
            ngroup = atoi(std::getenv("S2_NGROUP"));
        }
*/
        printf("%s:%d: batch_size: %ld, nchan: %ld, nlat_in: %ld, nlon_in: %ld, nlat_out: %ld, nlon_out: %ld, nrows: %ld, nnz_tot: %ld, K: %ld\n",
                __func__, __LINE__, batch_size, nchan, nlat_in, nlon_in, nlat_out, nlon_out, nrows, col_idx.size(0), K);

        // get stream
        auto stream = at::cuda::getCurrentCUDAStream().stream();

        // assert
        static_assert(0 == (ELXTH_MAX % 2));
#if 0
        // allocate output
        int64_t out_dims[] = {B, C, K, Ho, Wo};
        auto options = torch::TensorOptions().device(inp.device()).dtype(inp.dtype());
        torch::Tensor out = torch::zeros(out_dims, options);


        // pick the correct launch config
        if (Wo <= 64 * ELXTH_MAX) {
            AT_DISPATCH_FLOATING_TYPES(inp.scalar_type(), "disco_forward_cuda", ([&] {
                                           launch_kernel<64, 1, scalar_t>(
                                               BC, Hi, Wi, K, Ho, Wo, nrows, roff_idx.data_ptr<int64_t>(),
                                               ker_idx.data_ptr<int64_t>(), row_idx.data_ptr<int64_t>(),
                                               col_idx.data_ptr<int64_t>(), val.data_ptr<scalar_t>(),
                                               inp.data_ptr<scalar_t>(), out.data_ptr<scalar_t>(), stream);
                                       }));
        } else if (Wo <= 128 * ELXTH_MAX) {
            AT_DISPATCH_FLOATING_TYPES(inp.scalar_type(), "disco_forward_cuda", ([&] {
                                           launch_kernel<128, (ELXTH_MAX / 2) + 1, scalar_t>(
                                               BC, Hi, Wi, K, Ho, Wo, nrows, roff_idx.data_ptr<int64_t>(),
                                               ker_idx.data_ptr<int64_t>(), row_idx.data_ptr<int64_t>(),
                                               col_idx.data_ptr<int64_t>(), val.data_ptr<scalar_t>(),
                                               inp.data_ptr<scalar_t>(), out.data_ptr<scalar_t>(), stream);
                                       }));
        } else if (Wo <= 256 * ELXTH_MAX) {
            AT_DISPATCH_FLOATING_TYPES(inp.scalar_type(), "disco_forward_cuda", ([&] {
                                           launch_kernel<256, (ELXTH_MAX / 2) + 1, scalar_t>(
                                               BC, Hi, Wi, K, Ho, Wo, nrows, roff_idx.data_ptr<int64_t>(),
                                               ker_idx.data_ptr<int64_t>(), row_idx.data_ptr<int64_t>(),
                                               col_idx.data_ptr<int64_t>(), val.data_ptr<scalar_t>(),
                                               inp.data_ptr<scalar_t>(), out.data_ptr<scalar_t>(), stream);
                                       }));
        } else if (Wo <= 512 * ELXTH_MAX) {
            AT_DISPATCH_FLOATING_TYPES(inp.scalar_type(), "disco_forward_cuda", ([&] {
                                           launch_kernel<512, (ELXTH_MAX / 2) + 1, scalar_t>(
                                               BC, Hi, Wi, K, Ho, Wo, nrows, roff_idx.data_ptr<int64_t>(),
                                               ker_idx.data_ptr<int64_t>(), row_idx.data_ptr<int64_t>(),
                                               col_idx.data_ptr<int64_t>(), val.data_ptr<scalar_t>(),
                                               inp.data_ptr<scalar_t>(), out.data_ptr<scalar_t>(), stream);
                                       }));
        } else if (Wo <= 1024 * ELXTH_MAX) {
            AT_DISPATCH_FLOATING_TYPES(inp.scalar_type(), "disco_forward_cuda", ([&] {
                                           launch_kernel<1024, (ELXTH_MAX / 2) + 1, scalar_t>(
                                               BC, Hi, Wi, K, Ho, Wo, nrows, roff_idx.data_ptr<int64_t>(),
                                               ker_idx.data_ptr<int64_t>(), row_idx.data_ptr<int64_t>(),
                                               col_idx.data_ptr<int64_t>(), val.data_ptr<scalar_t>(),
                                               inp.data_ptr<scalar_t>(), out.data_ptr<scalar_t>(), stream);
                                       }));
        } else {
            fprintf(stderr, "%s:%d: error, unsupported Wo value (%ld), max supported is %d\n", __FILE__, __LINE__, Wo,
                    1024 * ELXTH_MAX);
            exit(EXIT_FAILURE);
        }
#else

#if 0 // FUSED VERSION
        // switch to channel-last
        // version with fused enisum 

        int64_t ngroup = weights.size(0);
        int64_t chan_x_grp_out = weights.size(1);
        int64_t chan_x_grp_in  = weights.size(2);
        int64_t weight_k = weights.size(3);

        int64_t nchan_out = ngroup*chan_x_grp_out;
        
        printf("weight tensor shape: %ld, %ld, %ld, %ld\n", ngroup, chan_x_grp_out, chan_x_grp_in, weight_k); fflush(stdout);

        if (nchan != chan_x_grp_in*ngroup || K != weight_k) {
            fprintf(stderr,
                    "%s:%d: error, dimension mismatch for weight tensor!\n",
                    __func__, __LINE__);
            exit(EXIT_FAILURE);
        }


        // input:  inp[B][Ci][Hi][Wi] -> inp[B][Hi][Wi][Ci]
        //
        // output: out[[B][Ho][Wo][Co] -> out[B][Co][Ho][Wo]
        //         with Co = ngroup*chan_x_grp_out


        // switch to channel-last

        // extract dtype
        auto x_type = inp.dtype();
        torch::Tensor xP = inp.to(torch::kFloat32);
    
        // exract memory format: this is much safer than checking is_contiguous(at::MemoryFormat::ChannelsLast)
        // the former fails for num_channels == 1
        bool x_is_channels_last = xP.strides()[1] == 1;

        // transpose if required
        if (!x_is_channels_last) { xP = permute_4D_to0231(xP); }

#if 1
        int64_t out_dims[] = {batch_size, nlat_out, nlon_out, nchan_out};
        auto options = torch::TensorOptions().device(inp.device()).dtype(inp.dtype());
        torch::Tensor yP = torch::zeros(out_dims, options); // this will be empty_like() 
        // y is {batch_size, nlat_out, nlon_out, nchan_out},
#else
        // to test before fusion
        int64_t out_dims[] = {batch_size, nlat_out, nlon_out, nchan*K};
        auto options = torch::TensorOptions().device(inp.device()).dtype(inp.dtype());
        torch::Tensor yP = torch::zeros(out_dims, options);
        // y is {batch_size, nlat_out, nlon_out, nchan*K},
#endif
        // call channel-last kernel implementation
        s2_disco_fwd_dispatch(batch_size,
                              nchan,
                              nchan_out,
                              ngroup,
                              nlon_in,
                              //nlat_in,
                              nlat_out,
                              nlon_out,
                              K,
                              xP,
                              roff_idx,
                              col_idx,
                              val,
                              weights,
                              yP);

#if 1
        // switch back to original layout;
        // I'm assuming that if x was passed as channel last, then
        // the output tensor should be K last
        //torch::Tensor y = yP;
        //if (!x_is_channels_last) { 
        //    y = permute_4D_to0312(y);
        //    // make y {batch_size, nchan_out, nlat_out, nlon_out}
        //}
#else
        // to test before fusion
        torch::Tensor y = yP;
        if (!x_is_channels_last) { 
            y = permute_4D_to0312(y);
            // make y {batch_size, nchan, K, nlat_out, nlon_out}
            y = y.reshape({batch_size, nchan, K, nlat_out, nlon_out});
        } else {
            // make y {batch_size, nlat_out, nlon_out, nchan, K}
            y = y.reshape({batch_size, nlat_out, nlon_out, nchan, K});
        }
#endif

#else // VERSION WITH SEPARATED EINSUM
        // switch to channel-last
        // version with fused enisum 

        auto x_type = inp.dtype();
        auto xP = inp.to(torch::kFloat32);

        // to test before fusion
        int64_t out_dims[] = {batch_size, nlat_out, nlon_out, nchan*K};
        //auto options = torch::TensorOptions().device(inp.device()).dtype(inp.dtype());
        torch::Tensor yP = torch::empty(out_dims, xP.options());

        // call channel-last kernel implementation
        s2_disco_fwd_dispatch(batch_size,
                              nchan,
                              nlon_in,
                              //nlat_in,
                              nlat_out,
                              nlon_out,
                              K,
                              xP,
                              roff_idx,
                              col_idx,
                              val,
                              yP);

        auto y = yP.to(x_type);
        
        torch::Tensor out = y;
#endif

#endif // closes ORIGINAL if
#if 1
        if (std::getenv("S2_DISCO_DUMP_Y")) {
            printf("waiting for kernel to finish...");
            CHECK_CUDA(hipStreamSynchronize(stream));
            printf("done\n");
            fflush(stdout);
            //dump_tensor("yout.txt", out);
            //dump_csr_linear("csr_disco.txt", roff_idx, ker_idx, row_idx, col_idx, val);
            //dump_out_kers("out_kers", out);
        }
#endif
        return out;
    }

    TORCH_LIBRARY_IMPL(disco_kernels, CUDA, m)
    {
        m.impl("forward",  &disco_cuda_fwd);
    }

    // utility functions
    void dump_tensor(const char *fname, at::Tensor t) {

            size_t n = 1;
            for(int i = 0; i < t.dim(); i++) {
                    n *= t.size(i);
            }

            float *data_h = (float *)malloc(sizeof(*data_h)*n);
            if (!data_h) {
                    fprintf(stderr, "Cannot allcoate %zu bytes!\n", sizeof(*data_h)*n);
                    exit(EXIT_FAILURE);
            }

            float *float_d = t.data_ptr<float>();

            CHECK_CUDA(hipMemcpy(data_h, float_d, sizeof(*data_h)*n, hipMemcpyDeviceToHost));

            printf("Writing data to file...");

            FILE *fp = fopen(fname, "w");
            if (!fp) {
                    fprintf(stderr, "Cannot open file %s for writing!\n", fname);
                    exit(EXIT_FAILURE);
            }

            for(size_t i = 0; i < n; i++) {
                    fprintf(fp, "%f\n", data_h[i]);
            }

            fclose(fp);
            printf("done\n");

            free(data_h);

            return;
    }

    void dump_csr(const char *fname,
                  at::Tensor roff,
                  at::Tensor cols) {

            int64_t nrows = roff.size(0)-1;
            int64_t nnz = cols.size(0);

            int64_t *roff_h = new int64_t[nrows+1];
            int64_t *cols_h = new int64_t[nnz];

            int64_t *roff_d = roff.data_ptr<int64_t>();
            int64_t *cols_d = cols.data_ptr<int64_t>();

            CHECK_CUDA(hipMemcpy(roff_h, roff_d, sizeof(*roff_h)*(nrows+1), hipMemcpyDeviceToHost));
            CHECK_CUDA(hipMemcpy(cols_h, cols_d, sizeof(*cols_d)*nnz      , hipMemcpyDeviceToHost));

            printf("Writing data to file...");

            FILE *fp = fopen(fname, "w");
            if (!fp) {
                    fprintf(stderr, "Cannot open file %s for writing!\n", fname);
                    exit(EXIT_FAILURE);
            }
            for(int64_t r = 0; r < nrows; r++) {

                    fprintf(fp, "%10ld %10ld", r, roff_h[r+1]-roff_h[r]);

                    for(int64_t o = roff_h[r]; o < roff_h[r+1]; o++) {
                            fprintf(fp, "%10ld", cols_h[o]);
                    }
                    fprintf(fp, "\n");
            }
            fclose(fp);
            printf("done\n");

            delete [] roff_h;
            delete [] cols_h;
    }
    
    void dump_csr_linear(const char *fname,
                         at::Tensor roff,
                         at::Tensor kers,
                         at::Tensor rows,
                         at::Tensor cols,
                         at::Tensor vals) {

            int64_t nrows = roff.size(0)-1;
            int64_t nnz = cols.size(0);

            int64_t *roff_h = new int64_t[nrows+1];
            int64_t *kers_h = new int64_t[nnz];
            int64_t *rows_h = new int64_t[nnz];
            int64_t *cols_h = new int64_t[nnz];
            float   *vals_h = new float[nnz];

            int64_t *roff_d = roff.data_ptr<int64_t>();
            int64_t *kers_d = kers.data_ptr<int64_t>();
            int64_t *rows_d = rows.data_ptr<int64_t>();
            int64_t *cols_d = cols.data_ptr<int64_t>();
            float   *vals_d = vals.data_ptr<float>();

            CHECK_CUDA(hipMemcpy(roff_h, roff_d, sizeof(*roff_h)*(nrows+1), hipMemcpyDeviceToHost));
            CHECK_CUDA(hipMemcpy(kers_h, kers_d, sizeof(*kers_h)*nnz      , hipMemcpyDeviceToHost));
            CHECK_CUDA(hipMemcpy(rows_h, rows_d, sizeof(*rows_h)*nnz      , hipMemcpyDeviceToHost));
            CHECK_CUDA(hipMemcpy(cols_h, cols_d, sizeof(*cols_h)*nnz      , hipMemcpyDeviceToHost));
            CHECK_CUDA(hipMemcpy(vals_h, vals_d, sizeof(*vals_h)*nnz      , hipMemcpyDeviceToHost));

            printf("Writing data to file...");

            FILE *fp = fopen(fname, "w");
            if (!fp) {
                    fprintf(stderr, "Cannot open file %s for writing!\n", fname);
                    exit(EXIT_FAILURE);
            }
            fprintf(fp, "COLS:\n");
            for(int64_t r = 0; r < nrows; r++) {

                    fprintf(fp, "%10ld %10ld", r, roff_h[r+1]-roff_h[r]);

                    for(int64_t o = roff_h[r]; o < roff_h[r+1]; o++) {
                            fprintf(fp, "%10ld", cols_h[o]);
                    }
                    fprintf(fp, "\n");
            }
            fprintf(fp, "KERS:\n");
            for(int64_t r = 0; r < nrows; r++) {

                    fprintf(fp, "%10ld %10ld", r, roff_h[r+1]-roff_h[r]);

                    for(int64_t o = roff_h[r]; o < roff_h[r+1]; o++) {
                            fprintf(fp, "%10ld", kers_h[o]);
                    }
                    fprintf(fp, "\n");
            }
            fprintf(fp, "ROWS:\n");
            for(int64_t r = 0; r < nrows; r++) {

                    fprintf(fp, "%10ld %10ld", r, roff_h[r+1]-roff_h[r]);

                    for(int64_t o = roff_h[r]; o < roff_h[r+1]; o++) {
                            fprintf(fp, "%10ld", rows_h[o]);
                    }
                    fprintf(fp, "\n");
            }
            fprintf(fp, "VALS:\n");
            for(int64_t r = 0; r < nrows; r++) {

                    fprintf(fp, "%10ld %10ld", r, roff_h[r+1]-roff_h[r]);

                    for(int64_t o = roff_h[r]; o < roff_h[r+1]; o++) {
                            fprintf(fp, "%10f", vals_h[o]);
                    }
                    fprintf(fp, "\n");
            }
            fclose(fp);
            printf("done\n");

            delete [] roff_h;
            delete [] kers_h;
            delete [] rows_h;
            delete [] cols_h;
            delete [] vals_h;
    }
}

