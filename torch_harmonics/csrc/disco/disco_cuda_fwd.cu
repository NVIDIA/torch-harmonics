#include "hip/hip_runtime.h"
// coding=utf-8
//
// SPDX-FileCopyrightText: Copyright (c) 2024 The torch-harmonics Authors. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
// list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
// this list of conditions and the following disclaimer in the documentation
// and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "disco.h"
#include "disco_cuda.cuh"


template<int BDIM_X,
         int ELXTH,
         typename REAL_T>
__device__ void disco_fwd_d(const int Hi,
                            const int Wi,
                            const int K,
                            const int Ho,
                            const int Wo,
                            const int pscale,
                            const int64_t *__restrict__ roff,
                            const int64_t *__restrict__ kers, 
                            const int64_t *__restrict__ rows,
                            const int64_t *__restrict__ cols,
                            const REAL_T  *__restrict__ vals, 
                            const REAL_T  *__restrict__ inp, 
                                  REAL_T  *__restrict__ out) {

  const int tid = threadIdx.x;
  
  const int64_t bidx = blockIdx.x; // gloabl row
  const int64_t bidy = blockIdx.y; // bc
  
  int64_t soff = roff[bidx];
  int64_t eoff = roff[bidx+1];
  
  const int64_t ker = kers[soff];
  const int64_t row = rows[soff];
  
  inp += bidy*Hi*Wi;
  out += bidy*K*Ho*Wo + ker*Ho*Wo + row*Wo;
  
  REAL_T __reg[ELXTH] = {0};
  
  // align to larger supported fp type
  extern __shared__ __align__(sizeof(double)) unsigned char __sh_ptr[]; // REAL_T __sh[2*Wi + ppscale*(BDIM_X*ELXTH - Wo)]
  REAL_T *__sh = reinterpret_cast<REAL_T *>(__sh_ptr);
  
  int col_prev = cols[soff];
  
  int h_prev = col_prev / Wi;
  int w_prev = col_prev % Wi;
  
  // copy current inp row in shmem
  for(int i = tid; i < Wi; i += BDIM_X) {
    const REAL_T v = inp[h_prev*Wi + i];
    __sh[     i] = v;
    __sh[Wi + i] = v;
  }
  // locations __sh[2*Wi : ppscale*(BDIM_X*ELXTH-Wo)] are not used
  __syncthreads();
  
  // loops along the colums of CTA's row
  for(int64_t nz = soff; nz < eoff; nz++) {
    
    const int    col = cols[nz];
    const REAL_T val = vals[nz];
    
    // if we are processing a nz with a col value
    // leading to a new row of inp then copy it
    // to shmem;
    // checks whether (h_prev < h) with:
    //  (col >= col_prev - (col_prev % Wi) + Wi)
    if (col >= col_prev-w_prev+Wi) { 
      
      col_prev = col;
      h_prev = col / Wi;
      w_prev = col % Wi;
      
      __syncthreads();
      for(int i = tid; i < Wi; i += BDIM_X) {
	const REAL_T v = inp[h_prev*Wi + i];
	__sh[     i] = v;
	__sh[Wi + i] = v;
      }
      __syncthreads();
    }
    
    const int w = w_prev + (col-col_prev);
    
    #pragma unroll
    for (int i = 0; i < ELXTH; i++) {
      
      const int pp = i*BDIM_X + tid;
      
      // original lines:
      //
      //   if (pp >= Wo) break;
      //   const int wpp = (w + pscale*pp) % Wi;
      //
      // value of (w + pscale*pp) < (Wi + (Wi/Wo)*Wo) = 2*Wi
      // so we can allocate twice the amount of shmem, 
      // replicate the current inp row and avoid the costly mod
      //
      // also, to avoid the conditional, sh can be extended to 
      // cover the maximum location accessed during this loop
      //
      // REAL_T __sh[2*Wi + ppscale*NUM_REM]
      //
      //   Wi + (Wi/Wo)*BDIM_X*ELXTH = (since BDIM_X*ELXTH >= Wo) =
      // = Wi + (Wi/Wo)*(Wo + (BDIM_X*ELXTH - Wo)) =
      // = 2*Wi + ppscale*NUM_REM
      //
      // with NUM_REM = BDIM_X*ELXTH - Wo
      
      const int wpp = w + pscale*pp;
      
      __reg[i] += val*__sh[wpp];
      
    }
  }
  
  #pragma unroll
  for (int i = 0; i < ELXTH; i++) {
    
    const int pp = i*BDIM_X + tid;
    if (pp >= Wo) break;
    
    out[pp] = __reg[i];
  }
  
  return;
}



template<int BDIM_X,
         int ELXTH,
         typename REAL_T>
__global__ __launch_bounds__(BDIM_X)
void disco_fwd_blk_k(const int Hi,
                     const int Wi,
                     const int K,
                     const int Ho,
                     const int Wo,
                     const int pscale,
                     const int64_t *__restrict__ roff,
                     const int64_t *__restrict__ kers, 
                     const int64_t *__restrict__ rows,
                     const int64_t *__restrict__ cols,
                     const REAL_T  *__restrict__ vals, 
                     const REAL_T  *__restrict__ inp, 
                           REAL_T  *__restrict__ out) {

  disco_fwd_d<BDIM_X, ELXTH>(Hi, Wi, K, Ho, Wo, pscale, roff, kers, rows, cols, vals, inp, out);
  
  return;
}


template<int NTH,
         int ELXTH,
         typename REAL_T>
static void launch_kernel(int BC,
                          int Hi,
                          int Wi,
                          int K,
                          int Ho,
                          int Wo,
                          int64_t nrows,
                          int64_t *roff_d,
                          int64_t *ker_d, 
                          int64_t *row_d,
                          int64_t *col_d,
                          REAL_T   *val_d, 
                          REAL_T   *inp_d, 
                          REAL_T   *out_d,
                          hipStream_t stream) {

  static_assert(sizeof(REAL_T) == 2 ||
		sizeof(REAL_T) == 4 ||
		sizeof(REAL_T) == 8);
  
  if constexpr(ELXTH <= ELXTH_MAX) {
      if (NTH*ELXTH >= Wo) {
	dim3 grid(nrows, BC);
        
	const int pscale = Wi/Wo;
	size_t shmem = sizeof(*out_d)*(Wi*2 + pscale*(NTH*ELXTH-Wo));
	
	disco_fwd_blk_k<NTH, ELXTH><<<grid, NTH, shmem, stream>>>(Hi, Wi,
								  K, Ho, Wo, pscale,
								  roff_d,
								  ker_d, row_d, col_d, val_d,
								  inp_d, out_d);
      } else {
	launch_kernel<NTH, ELXTH+1>(BC,
				    Hi, Wi, 
				    K, Ho, Wo,
				    nrows,
				    roff_d, 
				    ker_d, row_d, col_d, val_d,
				    inp_d, out_d,
				    stream);
      }
    }
  return;
}



torch::Tensor disco_cuda_fwd(torch::Tensor inp,
			     torch::Tensor roff_idx,
			     torch::Tensor ker_idx,
			     torch::Tensor row_idx,
			     torch::Tensor col_idx,
			     torch::Tensor val,
			     int64_t K,
			     int64_t Ho,
			     int64_t Wo) {
  
  // some sanity checks
  CHECK_CUDA_INPUT_TENSOR(inp);
  CHECK_CUDA_INPUT_TENSOR(roff_idx);
  CHECK_CUDA_INPUT_TENSOR(ker_idx);
  CHECK_CUDA_INPUT_TENSOR(row_idx);
  CHECK_CUDA_INPUT_TENSOR(col_idx);
  CHECK_CUDA_INPUT_TENSOR(val);

  // extract some shapes
  int64_t B = inp.size(0);
  int64_t C = inp.size(1);
  int64_t BC = B * C;
  int64_t Hi = inp.size(2);
  int64_t Wi = inp.size(3);
  int64_t nrows = roff_idx.size(0) - 1;

  // allocate output
  int64_t out_dims[] = {B, C, K, Ho, Wo};
  auto options = torch::TensorOptions().device(inp.device()).dtype(inp.dtype());
  torch::Tensor out = torch::zeros(out_dims, options);

  // get stream
  auto stream = at::cuda::getCurrentCUDAStream().stream();
  
  // assert
  static_assert(0 == (ELXTH_MAX%2));

  // pick the correct launch config
  if      (Wo <=   64*ELXTH_MAX) {
    AT_DISPATCH_FLOATING_TYPES(inp.scalar_type(), "disco_forward_cuda", ([&] {
	  launch_kernel<64, 1, scalar_t>(BC, Hi, Wi, K, Ho, Wo, nrows,
					 roff_idx.data_ptr<int64_t>(),
					 ker_idx.data_ptr<int64_t>(),
					 row_idx.data_ptr<int64_t>(),
					 col_idx.data_ptr<int64_t>(),
					 val.data_ptr<scalar_t>(),
					 inp.data_ptr<scalar_t>(),
					 out.data_ptr<scalar_t>(),
					 stream);
	    }));
  }
  else if (Wo <=  128*ELXTH_MAX) {
    AT_DISPATCH_FLOATING_TYPES(inp.scalar_type(), "disco_forward_cuda", ([&] {
	  launch_kernel<128, (ELXTH_MAX/2)+1, scalar_t>(BC, Hi, Wi, K, Ho, Wo, nrows,
							roff_idx.data_ptr<int64_t>(),
							ker_idx.data_ptr<int64_t>(),
							row_idx.data_ptr<int64_t>(),
							col_idx.data_ptr<int64_t>(),
							val.data_ptr<scalar_t>(),
							inp.data_ptr<scalar_t>(),
							out.data_ptr<scalar_t>(),
							stream);
	    }));
  }
  else if (Wo <=  256*ELXTH_MAX) {
    AT_DISPATCH_FLOATING_TYPES(inp.scalar_type(), "disco_forward_cuda", ([&] {
          launch_kernel<256, (ELXTH_MAX/2)+1, scalar_t>(BC, Hi, Wi, K, Ho, Wo, nrows,
                                                        roff_idx.data_ptr<int64_t>(),
                                                        ker_idx.data_ptr<int64_t>(),
                                                        row_idx.data_ptr<int64_t>(),
                                                        col_idx.data_ptr<int64_t>(),
                                                        val.data_ptr<scalar_t>(),
                                                        inp.data_ptr<scalar_t>(),
                                                        out.data_ptr<scalar_t>(),
                                                        stream);
	    }));
  }
  else if (Wo <=  512*ELXTH_MAX) {
    AT_DISPATCH_FLOATING_TYPES(inp.scalar_type(), "disco_forward_cuda", ([&] {
          launch_kernel<512, (ELXTH_MAX/2)+1, scalar_t>(BC, Hi, Wi, K, Ho, Wo, nrows,
                                                        roff_idx.data_ptr<int64_t>(),
                                                        ker_idx.data_ptr<int64_t>(),
                                                        row_idx.data_ptr<int64_t>(),
                                                        col_idx.data_ptr<int64_t>(),
                                                        val.data_ptr<scalar_t>(),
                                                        inp.data_ptr<scalar_t>(),
                                                        out.data_ptr<scalar_t>(),
                                                        stream);
            }));
  }
  else if (Wo <= 1024*ELXTH_MAX) {
    AT_DISPATCH_FLOATING_TYPES(inp.scalar_type(), "disco_forward_cuda", ([&] {
          launch_kernel<1024, (ELXTH_MAX/2)+1, scalar_t>(BC, Hi, Wi, K, Ho, Wo, nrows,
							 roff_idx.data_ptr<int64_t>(),
							 ker_idx.data_ptr<int64_t>(),
							 row_idx.data_ptr<int64_t>(),
							 col_idx.data_ptr<int64_t>(),
							 val.data_ptr<scalar_t>(),
							 inp.data_ptr<scalar_t>(),
							 out.data_ptr<scalar_t>(),
							 stream);
            }));
  }
  else {
    fprintf(stderr,
            "%s:%d: error, unsupported Wo value (%ld), max supported is %d\n",
            __FILE__, __LINE__, Wo, 1024*ELXTH_MAX);
    exit(EXIT_FAILURE);
  }
  
  
  return out;
}
