#include "hip/hip_runtime.h"
// coding=utf-8
//
// SPDX-FileCopyrightText: Copyright (c) 2025 The torch-harmonics Authors. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
// list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
// this list of conditions and the following disclaimer in the documentation
// and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "attention.cuh"
#include "c10/core/MemoryFormat.h"

#include <ATen/core/TensorAccessor.h>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/CUDAUtils.h>

#include <ctime>
#include <hipcub/hipcub.hpp>
#include <limits>

#ifndef WARP_SIZE
#define WARP_SIZE (32)
#endif
#ifndef FULL_MASK
#define FULL_MASK (0xFFFFFFFF)
#endif
#ifndef THREADS
#define THREADS (64)
#endif
#ifndef DIV_UP
#define DIV_UP(a, b) (((a) + ((b)-1)) / (b))
#endif
#ifndef CHECK_CUDA
#define CHECK_CUDA(call)                                                                                                 \
    {                                                                                                                    \
        hipError_t err = call;                                                                                          \
        if (hipSuccess != err) {                                                                                        \
            fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                                                                          \
        }                                                                                                                \
    }
#endif

#include <iostream>
#include <chrono>
#include <string>

class ScopeTimer
{
  public:
    explicit ScopeTimer(const std::string &label = "") :
        label_(label), start_(std::chrono::high_resolution_clock::now())
    {
    }

    ~ScopeTimer()
    {
        auto end = std::chrono::high_resolution_clock::now();
        auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start_);
        std::cout << label_ << "Elapsed time: " << elapsed.count() << " ms" << std::endl;
    }

  private:
    std::string label_;
    std::chrono::high_resolution_clock::time_point start_;
};

static __device__ float __warp_sum(float val)
{
#pragma unroll
    for (int i = WARP_SIZE / 2; i; i /= 2) { val += __shfl_xor_sync(FULL_MASK, val, i); }
    return val;
}

// easier to understand version of manual shfl_xor_sync, performance appears similar
static __device__ float __warp_sum_cub(float val)
{
    // use cub to reduce within a warp
    __shared__ typename hipcub::WarpReduce<float>::TempStorage temp_storage;

    // 1. Compute sum (initially only in lane 0)
    float sum = hipcub::WarpReduce<float>(temp_storage).Sum(val);
    // 2. Broadcast sum to all threads
    sum = __shfl_sync(0xFFFFFFFF, sum, 0);
    return sum;
}

// This kernel computes the backward pass for the S2 attention mechanism, using
// shared memory as a cache and one warp per output point, warp-parallel over
// channels, which should be layed out in the fastest dimension for coalesced
// memory access.
template <int BDIM_X>
__global__ __launch_bounds__(BDIM_X) void s2_attention_bwd_dkvq_kernel(
    int num_channels, int nlon_in, int nlat_out, int nlon_out,
    const torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> kx,
    const torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> vx,
    const torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> qy,
    const torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> dy,
    torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> dydk,
    torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> dydv,
    torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> dydq,
    const torch::PackedTensorAccessor64<int64_t, 1, torch::RestrictPtrTraits> psi_col_idx,
    const torch::PackedTensorAccessor64<int64_t, 1, torch::RestrictPtrTraits> psi_row_offset,
    const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> quad_weights)
{

    extern __shared__ float sh[];
    float *sh_alpha_k = sh + threadIdx.y * num_channels * 5;
    float *sh_alpha_vw = sh_alpha_k + num_channels;
    float *sh_alpha_kvw = sh_alpha_vw + num_channels;
    float *sh_dy = sh_alpha_kvw + num_channels;
    float *sh_qy = sh_dy + num_channels;
    // (optionally, could use more shared memory for other intermediates)

    const uint64_t batchId = blockIdx.y;
    const uint64_t wid = uint64_t(blockIdx.x) * blockDim.y + threadIdx.y;
    if (wid >= uint64_t(nlat_out) * nlon_in) return;
    const int tidx = threadIdx.x;
    const int ho = wid / nlon_out;
    const int wo = wid - (ho * nlon_out);

    // Zero shared memory
    for (int chan = tidx; chan < num_channels; chan += WARP_SIZE) {
        sh_alpha_k[chan] = 0.0f;
        sh_alpha_vw[chan] = 0.0f;
        sh_alpha_kvw[chan] = 0.0f;
        sh_dy[chan] = dy[batchId][chan][ho][wo];
        sh_qy[chan] = qy[batchId][chan][ho][wo];
    }
    float alpha_sum = 0.0f;
    float qdotk_max = -FLT_MAX;
    float integral = 0.0f;
    __syncthreads();

    const int64_t rbeg = psi_row_offset[ho];
    const int64_t rend = psi_row_offset[ho + 1];
    const int rlen = rend - rbeg;

    // 1st pass: accumulate alpha_sum, integral, and shared stats, along with a progressively computed qdotk_max.
    for (int off = 0; off < rlen; off++) {
        const int64_t col = psi_col_idx[rbeg + off];
        const int hi = col / nlon_in;
        const int wi = col - (hi * nlon_in);
        const int wip = (wi + wo) - ((wi + wo) / nlon_in) * nlon_in;
        float qdotk = 0.0f, gdotv = 0.0f;
        for (int chan = tidx; chan < num_channels; chan += WARP_SIZE) {
            qdotk += sh_qy[chan] * kx[batchId][chan][hi][wip];
            gdotv += sh_dy[chan] * vx[batchId][chan][hi][wip];
        }
        qdotk = __warp_sum_cub(qdotk);
        gdotv = __warp_sum_cub(gdotv);
        float qdotk_max_tmp = max(qdotk_max, qdotk);
        float alpha_inz = expf(qdotk - qdotk_max_tmp) * quad_weights[hi];
        float max_correction = expf(qdotk_max - qdotk_max_tmp);
        alpha_sum = alpha_sum * max_correction + alpha_inz;
        integral = integral * max_correction + alpha_inz * gdotv;
        for (int chan = tidx; chan < num_channels; chan += WARP_SIZE) {
            float kxval = kx[batchId][chan][hi][wip];
            sh_alpha_k[chan] = sh_alpha_k[chan] * max_correction + alpha_inz * kxval;
            sh_alpha_vw[chan] = sh_alpha_vw[chan] * max_correction + alpha_inz * gdotv;
            sh_alpha_kvw[chan] = sh_alpha_kvw[chan] * max_correction + alpha_inz * kxval * gdotv;
        }
        qdotk_max = qdotk_max_tmp;
    }

    integral /= alpha_sum;

    // Write dydq
    for (int chan = tidx; chan < num_channels; chan += WARP_SIZE) {
        dydq[batchId][chan][ho][wo]
            = (sh_alpha_kvw[chan] * alpha_sum - sh_alpha_vw[chan] * sh_alpha_k[chan]) / (alpha_sum * alpha_sum);
    }

    // Third pass: accumulate gradients for k and v
    for (int off = 0; off < rlen; off++) {
        const int64_t col = psi_col_idx[rbeg + off];
        const int hi = col / nlon_in;
        const int wi = col - (hi * nlon_in);
        const int wip = (wi + wo) - ((wi + wo) / nlon_in) * nlon_in;
        float qdotk = 0.0f, gdotv = 0.0f;
        for (int chan = tidx; chan < num_channels; chan += WARP_SIZE) {
            qdotk += qy[batchId][chan][ho][wo] * kx[batchId][chan][hi][wip];
            gdotv += sh_dy[chan] * vx[batchId][chan][hi][wip];
        }
        qdotk = __warp_sum_cub(qdotk);
        gdotv = __warp_sum_cub(gdotv);
        float alpha_inz = expf(qdotk - qdotk_max) * quad_weights[hi];
        for (int chan = tidx; chan < num_channels; chan += WARP_SIZE) {
            float qyval = qy[batchId][chan][ho][wo];
            float dyval = sh_dy[chan];
            atomicAdd(&dydk[batchId][chan][hi][wip], qyval * (alpha_inz / alpha_sum) * (gdotv - integral));
            atomicAdd(&dydv[batchId][chan][hi][wip], (alpha_inz / alpha_sum) * dyval);
        }
    }
}

std::tuple<at::Tensor, at::Tensor, at::Tensor> s2_attention_bwd_dkvq_cuda(at::Tensor kx, at::Tensor vx, at::Tensor qy,
                                                                          at::Tensor dy, at::Tensor quad_weights,
                                                                          at::Tensor psi_col_idx, at::Tensor psi_row_off,
                                                                          int nlon_in, int nlat_out, int nlon_out)
{

    CHECK_CUDA_TENSOR(kx);
    CHECK_CUDA_TENSOR(vx);
    CHECK_CUDA_TENSOR(qy);
    CHECK_CUDA_TENSOR(quad_weights);
    CHECK_CUDA_TENSOR(psi_col_idx);
    CHECK_CUDA_TENSOR(psi_row_off);
    CHECK_CUDA_TENSOR(dy);

    auto stream = at::cuda::getCurrentCUDAStream().stream();

    auto k_channel_first = kx.strides()[1] == 1;
    auto v_channel_first = vx.strides()[1] == 1;
    auto q_channel_first = qy.strides()[1] == 1;
    auto dy_channel_first = dy.strides()[1] == 1;

    // Transpose to [batch, ho, wo, channel]
    nvtxRangePush("s2_attention_bwd_dkvq_kernel_mbT permute inputs");
    // auto* permute_timer = new ScopeTimer("permute inputs");

    // Permute kx,vx,qy,dy to [batch, ho, wo, channel] in memory layout, but keep the original shape [batch, channel, ho, wo]
    auto kxP = at::Tensor();
    if (!k_channel_first) {
        // printf("Permuting kx from [batch, channel, ho, wo] to [batch, ho, wo, channel]\n");
        kxP = kx.permute({0, 2, 3, 1}).contiguous().permute({0, 3, 1, 2});
    } else {
        kxP = kx;
    }
    auto vxP = at::Tensor();
    if (!v_channel_first) {
        // printf("Permuting vx from [batch, channel, ho, wo] to [batch, ho, wo, channel]\n");
        vxP = vx.permute({0, 2, 3, 1}).contiguous().permute({0, 3, 1, 2});
    } else {
        vxP = vx;
    }
    auto qyP = at::Tensor();
    if (!q_channel_first) {
        // printf("Permuting qy from [batch, channel, ho, wo] to [batch, ho, wo, channel]\n");
        qyP = qy.permute({0, 2, 3, 1}).contiguous().permute({0, 3, 1, 2});
    } else {
        qyP = qy;
    }
    auto dyP = at::Tensor();
    if (!dy_channel_first) {
        // printf("Permuting dy from [batch, channel, ho, wo] to [batch, ho, wo, channel]\n");
        dyP = dy.permute({0, 2, 3, 1}).contiguous().permute({0, 3, 1, 2});
    } else {
        dyP = dy;
    }
    // hipDeviceSynchronize();
    // delete permute_timer;
    nvtxRangePop();

    nvtxRangePush("s2_attention_bwd_dkvq_kernel_mbT output allocation & zero");
    auto dydk = torch::zeros_like(qyP);
    auto dydv = torch::zeros_like(qyP);
    auto dydq = torch::zeros_like(qyP);
    // print strdie of dydkP, dydvP, dydqP
    nvtxRangePop();

    size_t uo_num_channels = kx.size(1);
    const int batch_size = kx.size(0);

    dim3 block(WARP_SIZE, THREADS / WARP_SIZE);
    dim3 grid(DIV_UP(nlat_out * nlon_out, block.y), batch_size);
    size_t shared_size = sizeof(float) * uo_num_channels * 5 * block.y; // 4 arrays per warp

    hipEvent_t start, stop;
    float milliseconds = 0;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start, stream));

    s2_attention_bwd_dkvq_kernel<THREADS><<<grid, block, shared_size, stream>>>(
        uo_num_channels, nlon_in, nlat_out, nlon_out, kxP.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
        vxP.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
        qyP.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
        dyP.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
        dydk.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
        dydv.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
        dydq.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
        psi_col_idx.packed_accessor64<int64_t, 1, torch::RestrictPtrTraits>(),
        psi_row_off.packed_accessor64<int64_t, 1, torch::RestrictPtrTraits>(),
        quad_weights.packed_accessor32<float, 1, torch::RestrictPtrTraits>());

    CHECK_CUDA(hipEventRecord(stop, stream));
    CHECK_CUDA(hipEventSynchronize(stop));
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));

    // [1, 256, 1, (721, 1440), (721, 1440), "equiangular", "equiangular", 1e-5, 1e-5],
    // s2_attention_bwd_kernel execution time: 50.724865 ms
    // [1, 256, 1, (361, 720), (361, 720), "equiangular", "equiangular", 1e-5, 1e-5],
    // s2_attention_bwd_kernel execution time: 11.679744 ms
    // printf("s2_attention_bwd_kernel execution time: %f ms\n", milliseconds);
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    C10_CUDA_KERNEL_LAUNCH_CHECK();

    // Permute outputs back to memory layout given by input. if input had channels
    // first, leave it in that layout, otherwise permute layout back to [batch,
    // channel, ho, wo]
    if (!k_channel_first) dydk = dydk.contiguous();
    if (!v_channel_first) dydv = dydv.contiguous();
    if (!q_channel_first) dydq = dydq.contiguous();

    // printf("dydk strides:[");
    // for(auto& stride : dydk.strides()) {
    //   printf("%ld,", stride);
    // }
    // printf("]\n");
    // hipDeviceSynchronize();
    // delete permute_output_timer;
    // nvtxRangePop();
    return std::make_tuple(dydk, dydv, dydq);
}
