#include "hip/hip_runtime.h"
// coding=utf-8
//
// SPDX-FileCopyrightText: Copyright (c) 2025 The torch-harmonics Authors. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
// list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
// this list of conditions and the following disclaimer in the documentation
// and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "attention.cuh"
#include "c10/core/MemoryFormat.h"

#include <ATen/core/TensorAccessor.h>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/CUDAUtils.h>

#include <ctime>
#include <hipcub/hipcub.hpp>
#include <limits>

#ifndef WARP_SIZE
#define WARP_SIZE (32)
#endif
#ifndef FULL_MASK
#define FULL_MASK (0xFFFFFFFF)
#endif
#ifndef THREADS
#define THREADS (64)
#endif
#ifndef DIV_UP
#define DIV_UP(a,b) (((a)+((b)-1))/(b))
#endif
#ifndef CHECK_CUDA
#define CHECK_CUDA(call) {                                            \
    hipError_t err = call;                                           \
    if( hipSuccess != err) {                                         \
      fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\\n",  \
              __FILE__, __LINE__, hipGetErrorString( err) );         \
      exit(EXIT_FAILURE);                                             \
    }}
#endif

#include <iostream>
#include <chrono>
#include <string>

class ScopeTimer {
public:
  explicit ScopeTimer(const std::string& label = "")
    : label_(label), start_(std::chrono::high_resolution_clock::now()) {}

  ~ScopeTimer() {
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start_);
    std::cout << label_ << "Elapsed time: " << elapsed.count() << " ms" << std::endl;
  }

private:
  std::string label_;
  std::chrono::high_resolution_clock::time_point start_;
};

static __device__ float __warp_sum(float val) {
#pragma unroll
  for(int i = WARP_SIZE/2; i; i /= 2) {
    val += __shfl_xor_sync(FULL_MASK, val, i);
  }
  return val;

}

// easier to understand version of manual shfl_xor_sync, performance appears similar
static __device__ float __warp_sum_cub(float val) {
  // use cub to reduce within a warp
  __shared__ typename hipcub::WarpReduce<float>::TempStorage temp_storage;
  
  // 1. Compute sum (initially only in lane 0)
  float sum = hipcub::WarpReduce<float>(temp_storage).Sum(val);
  // 2. Broadcast sum to all threads
  sum = __shfl_sync(0xFFFFFFFF, sum, 0);
  return sum;
}

// This kernel computes the backward pass for the S2 attention mechanism, using
// shared memory as a cache and one warp per output point, warp-parallel over
// channels, which should be layed out in the fastest dimension for coalesced
// memory access.
template<int BDIM_X>
__global__
__launch_bounds__(BDIM_X)
  void s2_attention_bwd_dkvq_kernel(
                                        int num_channels,
                                        int nlon_in,
                                        int nlat_out,
                                        int nlon_out,
                                        const torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> kx,
                                        const torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> vx,
                                        const torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> qy,
                                        const torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> dy,
                                        torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> dydk,
                                        torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> dydv,
                                        torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> dydq,
                                        const torch::PackedTensorAccessor64<int64_t, 1, torch::RestrictPtrTraits> psi_col_idx,
                                        const torch::PackedTensorAccessor64<int64_t, 1, torch::RestrictPtrTraits> psi_row_offset,
                                        const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> quad_weights) {

  extern __shared__ float sh[];
  float* sh_alpha_k = sh + threadIdx.y * num_channels * 5;
  float* sh_alpha_vw = sh_alpha_k + num_channels;
  float* sh_alpha_kvw = sh_alpha_vw + num_channels;
  float *sh_dy = sh_alpha_kvw + num_channels;
  float* sh_qy = sh_dy + num_channels;
  // (optionally, could use more shared memory for other intermediates)

  const uint64_t batchId = blockIdx.y;
  const uint64_t wid = uint64_t(blockIdx.x) * blockDim.y + threadIdx.y;
  if (wid >= uint64_t(nlat_out) * nlon_in) return;
  const int tidx = threadIdx.x;
  const int ho = wid / nlon_out;
  const int wo = wid - (ho * nlon_out);

  // Zero shared memory
  for (int chan = tidx; chan < num_channels; chan += WARP_SIZE) {
    sh_alpha_k[chan] = 0.0f;
    sh_alpha_vw[chan] = 0.0f;
    sh_alpha_kvw[chan] = 0.0f;
    sh_dy[chan] = dy[batchId][chan][ho][wo];
    sh_qy[chan] = qy[batchId][chan][ho][wo];
  }
  float alpha_sum = 0.0f;
  float qdotk_max = -FLT_MAX;
  float integral = 0.0f;
  __syncthreads();

  const int64_t rbeg = psi_row_offset[ho];
  const int64_t rend = psi_row_offset[ho+1];
  const int rlen = rend - rbeg;

  // First pass: find qdotk_max
  for (int off = 0; off < rlen; off++) {
    const int64_t col = psi_col_idx[rbeg + off];
    const int hi = col / nlon_in;
    const int wi = col - (hi * nlon_in);
    const int wip = (wi + wo) - ((wi + wo) / nlon_in) * nlon_in;
    float qdotk = 0.0f;
    for (int chan = tidx; chan < num_channels; chan += WARP_SIZE) {
      qdotk += sh_qy[chan] * kx[batchId][chan][hi][wip];
    }
    qdotk = __warp_sum_cub(qdotk);
    qdotk_max = max(qdotk_max, qdotk);
  }

  // Second pass: accumulate alpha_sum, integral, and shared stats
  for (int off = 0; off < rlen; off++) {
    const int64_t col = psi_col_idx[rbeg + off];
    const int hi = col / nlon_in;
    const int wi = col - (hi * nlon_in);
    const int wip = (wi + wo) - ((wi + wo) / nlon_in) * nlon_in;
    float qdotk = 0.0f, gdotv = 0.0f;
    for (int chan = tidx; chan < num_channels; chan += WARP_SIZE) {
      qdotk += sh_qy[chan] * kx[batchId][chan][hi][wip];
      gdotv += sh_dy[chan] * vx[batchId][chan][hi][wip];
    }
    qdotk = __warp_sum_cub(qdotk);
    gdotv = __warp_sum_cub(gdotv);
    float alpha_inz = expf(qdotk - qdotk_max) * quad_weights[hi];
    alpha_sum += alpha_inz;
    integral += alpha_inz * gdotv;
    for (int chan = tidx; chan < num_channels; chan += WARP_SIZE) {
      float kxval = kx[batchId][chan][hi][wip];
      sh_alpha_k[chan] += alpha_inz * kxval;
      sh_alpha_vw[chan] += alpha_inz * gdotv;
      sh_alpha_kvw[chan] += alpha_inz * kxval * gdotv;
    }
  }

  integral /= alpha_sum;

  // Write dydq
  for (int chan = tidx; chan < num_channels; chan += WARP_SIZE) {
    dydq[batchId][chan][ho][wo] = (sh_alpha_kvw[chan] * alpha_sum - sh_alpha_vw[chan] * sh_alpha_k[chan]) / (alpha_sum * alpha_sum);
  }

  // Third pass: accumulate gradients for k and v
  for (int off = 0; off < rlen; off++) {
    const int64_t col = psi_col_idx[rbeg + off];
    const int hi = col / nlon_in;
    const int wi = col - (hi * nlon_in);
    const int wip = (wi + wo) - ((wi + wo) / nlon_in) * nlon_in;
    float qdotk = 0.0f, gdotv = 0.0f;
    for (int chan = tidx; chan < num_channels; chan += WARP_SIZE) {
      qdotk += qy[batchId][chan][ho][wo] * kx[batchId][chan][hi][wip];
      gdotv += sh_dy[chan] * vx[batchId][chan][hi][wip];
    }
    qdotk = __warp_sum_cub(qdotk);
    gdotv = __warp_sum_cub(gdotv);
    float alpha_inz = expf(qdotk - qdotk_max) * quad_weights[hi];
    for (int chan = tidx; chan < num_channels; chan += WARP_SIZE) {
      float qyval = qy[batchId][chan][ho][wo];
      float dyval = sh_dy[chan];
      atomicAdd(&dydk[batchId][chan][hi][wip], qyval * (alpha_inz / alpha_sum) * (gdotv - integral));
      atomicAdd(&dydv[batchId][chan][hi][wip], (alpha_inz / alpha_sum) * dyval);
    }
  }
}




std::tuple<at::Tensor,at::Tensor,at::Tensor> s2_attention_bwd_dkvq_cuda(at::Tensor kx, at::Tensor vx,
                                                                        at::Tensor qy,
                                                                        at::Tensor dy,
                                                                        at::Tensor quad_weights,
                                                                        at::Tensor psi_col_idx,
                                                                        at::Tensor psi_row_off,
                                                                        int nlon_in, int nlat_out, int nlon_out) {

  CHECK_CUDA_TENSOR(kx);
  CHECK_CUDA_TENSOR(vx);
  CHECK_CUDA_TENSOR(qy);
  CHECK_CUDA_TENSOR(quad_weights);
  CHECK_CUDA_TENSOR(psi_col_idx);
  CHECK_CUDA_TENSOR(psi_row_off);
  CHECK_CUDA_TENSOR(dy);

  auto stream = at::cuda::getCurrentCUDAStream().stream();

  // Transpose to [batch, ho, wo, channel]
  nvtxRangePush("s2_attention_bwd_dkvq_kernel_mbT permute inputs");
  // auto* permute_timer = new ScopeTimer("permute inputs");

  // extract dtype
  auto kx_type = kx.dtype();
  auto vx_type = vx.dtype();
  auto qy_type = qy.dtype();
  auto dy_type = dy.dtype();

  // exract memory format
  auto kx_is_channels_last = kx.is_contiguous(at::MemoryFormat::Channels_last);
  auto vx_is_channels_last = vx.is_contiguous(at::MemoryFormat::Channels_last);
  auto qy_is_channels_last = qy.is_contiguous(at::MemoryFormat::Channels_last);
  auto dy_is_channels_last = dy.is_contiguous(at::MemoryFormat::Channels_last);

  // convert to channels-last
  auto kxP = kx.to(torch::kFloat32, at::MemoryFormat::ChannelsLast);
  auto vxP = vx.to(torch::kFloat32, at::MemoryFormat::ChannelsLast);
  auto qyP = qy.to(torch::kFloat32, at::MemoryFormat::ChannelsLast);
  auto dyP = dy.to(torch::kFloat32, at::MemoryFormat::ChannelsLast);

  // hipDeviceSynchronize();
  // delete permute_timer;
  nvtxRangePop();

  nvtxRangePush("s2_attention_bwd_dkvq_kernel_mbT output allocation & zero");
  auto dydk = torch::zeros_like(qyP);
  auto dydv = torch::zeros_like(qyP);
  auto dydq = torch::zeros_like(qyP);
  // print strdie of dydkP, dydvP, dydqP
  nvtxRangePop();

  size_t uo_num_channels = kx.size(1);
  const int batch_size = kx.size(0);

  dim3 block(WARP_SIZE, THREADS/WARP_SIZE);
  dim3 grid(DIV_UP(nlat_out*nlon_out, block.y), batch_size);
  size_t shared_size = sizeof(float) * uo_num_channels * 5 * block.y; // 4 arrays per warp

  hipEvent_t start, stop;
  float milliseconds = 0;
  CHECK_CUDA(hipEventCreate(&start));
  CHECK_CUDA(hipEventCreate(&stop));
  CHECK_CUDA(hipEventRecord(start, stream));

  s2_attention_bwd_dkvq_kernel<THREADS><<<
    grid, block, shared_size, stream>>>(
                                        uo_num_channels, nlon_in, nlat_out, nlon_out,
                                        kxP.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
                                        vxP.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
                                        qyP.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
                                        dyP.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
                                        dydk.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
                                        dydv.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
                                        dydq.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
                                        psi_col_idx.packed_accessor64<int64_t, 1, torch::RestrictPtrTraits>(),
                                        psi_row_off.packed_accessor64<int64_t, 1, torch::RestrictPtrTraits>(),
                                        quad_weights.packed_accessor32<float, 1, torch::RestrictPtrTraits>());
  
  CHECK_CUDA(hipEventRecord(stop, stream));
  CHECK_CUDA(hipEventSynchronize(stop));
  CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));

  // [1, 256, 1, (721, 1440), (721, 1440), "equiangular", "equiangular", 1e-5, 1e-5],
  // s2_attention_bwd_kernel_mbT execution time: 63.280128 ms
  // printf("s2_attention_bwd_kernel_mbT execution time: %f ms\n", milliseconds);
  CHECK_CUDA(hipEventDestroy(start));
  CHECK_CUDA(hipEventDestroy(stop));
  
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  // Permute outputs back to memory layout given by input. if input had channels
  // first, leave it in that layout, otherwise permute layout back to [batch,
  // channel, ho, wo]

  // convert back to original dtype
  dydk = dydk.to(kx_type);
  dydv = dydv.to(vx_type);
  dydq = dydq.to(qy_type);

  // permute back to original layout
  if(!kx_is_channels_last){
    dydk = dydk.to(kx_type, at::MemoryFormat::Contiguous);
  } else {
    dydk = dydk.to(kx_type);
  }
  if(!vx_is_channels_last){
    dydv = dydv.to(vx_type, at::MemoryFormat::Contiguous);
  } else {
    dydv = dydv.to(vx_type);
  }
  if(!qy_is_channels_last) {
    dydq = dydq.to(qy_type, at::MemoryFormat::Contiguous);
  } else {
    dydq = dydq.to(qy_type)
  }

  // printf("dydk strides:  [");
  // for(auto& stride : dydk.strides()) {
  //   printf("%ld,", stride);
  // }
  // printf("]\n");
  // hipDeviceSynchronize();
  // delete permute_output_timer;
  // nvtxRangePop();
  return std::make_tuple(dydk, dydv, dydq);

}

