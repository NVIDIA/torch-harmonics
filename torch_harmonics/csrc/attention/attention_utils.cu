#include "hip/hip_runtime.h"
// coding=utf-8
//
// SPDX-FileCopyrightText: Copyright (c) 2025 The torch-harmonics Authors. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
// list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
// this list of conditions and the following disclaimer in the documentation
// and/or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "attention.cuh"
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/CUDAUtils.h>

#include <hip/hip_runtime.h>

#include <hipcub/hipcub.hpp>
#include <limits>

#include "cudamacro.h"
#include "attention_utils.cuh"

#define THREADS (64)

#define TRANSP_WARPS_X_TILE_GENERIC (32)
#define TRANSP_WARPS_X_TILE_SM100    (4)

// BEGIN - CSR rows sorting kernels and functions
__global__ void set_rlen_rids_k(const int n,
                                const int64_t *__restrict__ offs,
                                      int *__restrict__ rids,
                                      int *__restrict__ rlen) {

    const int nth = gridDim.x*blockDim.x;
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;

    for(int i = tid; i < n; i += nth) {
        rids[i] = i;
        rlen[i] = offs[i+1]-offs[i];
    }

    return;
}   

at::Tensor sortRows(int nlat_out, at::Tensor row_off, hipStream_t stream) {

    int64_t *_row_off_d = reinterpret_cast<int64_t *>(row_off.data_ptr());

    auto options = torch::TensorOptions().dtype(torch::kInt32).device(row_off.device());

    torch::Tensor rids_d = torch::empty({nlat_out}, options);
    torch::Tensor rlen_d = torch::empty({nlat_out}, options);

    int *_rids_d = reinterpret_cast<int *>(rids_d.data_ptr());
    int *_rlen_d = reinterpret_cast<int *>(rlen_d.data_ptr());

    const int grid = DIV_UP(nlat_out, THREADS);
    const int block = THREADS;

    set_rlen_rids_k<<<grid, block, 0, stream>>>(nlat_out,
                                                _row_off_d,
                                                _rids_d,
                                                _rlen_d);

    torch::Tensor rids_sort_d = torch::empty({nlat_out}, options);
    torch::Tensor rlen_sort_d = torch::empty({nlat_out}, options);

    int *_rids_sort_d = reinterpret_cast<int *>(rids_sort_d.data_ptr());
    int *_rlen_sort_d = reinterpret_cast<int *>(rlen_sort_d.data_ptr());

    size_t temp_storage_bytes = 0;
    CHECK_CUDA(hipcub::DeviceRadixSort::SortPairsDescending(NULL, temp_storage_bytes,
                                                         _rlen_d, _rlen_sort_d, 
                                                         _rids_d, _rids_sort_d,
                                                         nlat_out, 0, sizeof(*_rlen_d)*8, stream));

    options = torch::TensorOptions().dtype(torch::kByte).device(row_off.device());
    torch::Tensor temp_storage_d = torch::empty({int64_t(temp_storage_bytes)}, options);

    void *_temp_storage_d = reinterpret_cast<void *>(temp_storage_d.data_ptr());

    CHECK_CUDA(hipcub::DeviceRadixSort::SortPairsDescending(_temp_storage_d, temp_storage_bytes,
                                                         _rlen_d, _rlen_sort_d, 
                                                         _rids_d, _rids_sort_d,
                                                         nlat_out, 0, sizeof(*_rlen_d)*8, stream));
    return rids_sort_d;
}
// END - CSR rows sorting kernels and functions


// BEGIN - 4D tensor permutation kernels and functions
template<int BDIM_X,
         int BDIM_Y,
         typename VAL_T>
__global__
__launch_bounds__(BDIM_X*BDIM_Y)
void  permute_to0231_k(const int nchn,
                       const int nlat,
                       const int nlon,
                       const torch::PackedTensorAccessor32<VAL_T, 4, torch::RestrictPtrTraits> src,
                             torch::PackedTensorAccessor32<VAL_T, 4, torch::RestrictPtrTraits> dst) {

    static_assert(!(BDIM_X & (BDIM_X-1)));
    static_assert(!(BDIM_Y & (BDIM_Y-1)));
    static_assert(BDIM_X >= BDIM_Y);

    __shared__ VAL_T sh[BDIM_X][BDIM_X+1];

    const int tidx = threadIdx.x;
    const int tidy = threadIdx.y;

    const int coff  = blockIdx.x*BDIM_X;           // channel offset
    const int woff  = blockIdx.y*BDIM_X;           // width offset
    const int batch = blockIdx.z / nlat;           // batch (same for all block)
    const int h     = blockIdx.z - (batch * nlat); // height (same for all block)

    const int nchn_full = (nchn-coff) >= BDIM_X;
    const int nlon_full = (nlon-woff) >= BDIM_X;

    if (nchn_full && nlon_full) {
        #pragma unroll
        for(int j = 0; j < BDIM_X; j += BDIM_Y) {
            sh[j+tidy][tidx] = src[batch][coff + j+tidy][h][woff+tidx];
        }
        __syncthreads();

        #pragma unroll
        for(int j = 0; j < BDIM_X; j += BDIM_Y) {
            dst[batch][h][woff + j+tidy][coff+tidx] = sh[tidx][j+tidy];
        }
    } else {
        if (woff+tidx < nlon) {
            #pragma unroll
            for(int j = 0; j < BDIM_X; j += BDIM_Y) {
                sh[j+tidy][tidx] = (coff + j+tidy < nchn) ? src[batch][coff + j+tidy][h][woff+tidx] : 0.f;
            }
        }
        __syncthreads();

        if (coff+tidx < nchn) {
            #pragma unroll
            for(int j = 0; j < BDIM_X; j += BDIM_Y) {
                if (woff + j+tidy < nlon) {
                    dst[batch][h][woff + j+tidy][coff+tidx] = sh[tidx][j+tidy];
                }
            }
        }
    }
    return;
}

__global__ void empty_k() {}

static int getPtxver() {
    hipFuncAttributes attrs;
    CHECK_CUDA(hipFuncGetAttributes(&attrs, reinterpret_cast<const void*>(empty_k)));
    return attrs.ptxVersion*10;
}

at::Tensor permute_4D_floatT_to0231(at::Tensor src, hipStream_t stream) {

    dim3 block;
    dim3 grid;

    block.x = WARP_SIZE;
    grid.x = DIV_UP(src.size(1), block.x);
    grid.y = DIV_UP(src.size(3), block.x);
    grid.z = src.size(2)*src.size(0);

    assert(grid.y < 65536);
    assert(grid.z < 65536);

    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(src.device());
    torch::Tensor dst = torch::empty({src.size(0), src.size(2), src.size(3), src.size(1)}, options);

    const int ptxv = getPtxver();

    // to be further specialized for additional archs, if necessary
    if (ptxv < 100) {
        block.y = TRANSP_WARPS_X_TILE_GENERIC;
        permute_to0231_k<WARP_SIZE, TRANSP_WARPS_X_TILE_GENERIC>
                        <<<grid, block, 0, stream>>>(src.size(1),
                                                     src.size(2),
                                                     src.size(3),
                                                     src.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
                                                     dst.packed_accessor32<float, 4, torch::RestrictPtrTraits>());
        CHECK_ERROR("permute_to0231_k_tile_generic");
    } else {
        block.y = TRANSP_WARPS_X_TILE_SM100;
        permute_to0231_k<WARP_SIZE, TRANSP_WARPS_X_TILE_SM100>
                        <<<grid, block, 0, stream>>>(src.size(1),
                                                     src.size(2),
                                                     src.size(3),
                                                     src.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
                                                     dst.packed_accessor32<float, 4, torch::RestrictPtrTraits>());
        CHECK_ERROR("permute_to0231_k_tile_sm100");
    }

    return dst;
}

template<int BDIM_X,
         int BDIM_Y,
         typename VAL_T>
__global__
__launch_bounds__(BDIM_X*BDIM_Y)
void  permute_to0312_k(const int nchn,
                       const int nlat,
                       const int nlon,
                       const torch::PackedTensorAccessor32<VAL_T, 4, torch::RestrictPtrTraits> src,
                             torch::PackedTensorAccessor32<VAL_T, 4, torch::RestrictPtrTraits> dst) {

    static_assert(!(BDIM_X & (BDIM_X-1)));
    static_assert(!(BDIM_Y & (BDIM_Y-1)));
    static_assert(BDIM_X >= BDIM_Y);

    __shared__ VAL_T sh[BDIM_X][BDIM_X+1];

    const int tidx = threadIdx.x;
    const int tidy = threadIdx.y;

    const int woff  = blockIdx.x*BDIM_X;           // width offset
    const int coff  = blockIdx.y*BDIM_X;           // channel offset
    const int batch = blockIdx.z / nlat;           // batch (same for all block)
    const int h     = blockIdx.z - (batch * nlat); // height (same for all block)

    const int nchn_full = (nchn-coff) >= BDIM_X;
    const int nlon_full = (nlon-woff) >= BDIM_X;

    if (nchn_full && nlon_full) {
        #pragma unroll
        for(int j = 0; j < BDIM_X; j += BDIM_Y) {
            sh[j+tidy][tidx] = src[batch][h][woff + j+tidy][coff+tidx];
        }
        __syncthreads();

        #pragma unroll
        for(int j = 0; j < BDIM_X; j += BDIM_Y) {
            dst[batch][coff + j+tidy][h][woff+tidx] = sh[tidx][j+tidy];
        }
    } else {
        if (coff+tidx < nchn) {
            #pragma unroll
            for(int j = 0; j < BDIM_X; j += BDIM_Y) {
                sh[j+tidy][tidx] = (woff + j+tidy < nlon) ? src[batch][h][woff + j+tidy][coff+tidx] : 0.f;
            }
        }
        __syncthreads();

        if (woff+tidx < nlon) {
            #pragma unroll
            for(int j = 0; j < BDIM_X; j += BDIM_Y) {
                if (coff + j+tidy < nchn) {
                    dst[batch][coff + j+tidy][h][woff+tidx] = sh[tidx][j+tidy];;
                }
            }
        }
    }
    return;
}

at::Tensor permute_4D_floatT_to0312(at::Tensor src, hipStream_t stream) {

    dim3 block;
    dim3 grid;

    block.x = WARP_SIZE;
    grid.x = DIV_UP(src.size(2), block.x);
    grid.y = DIV_UP(src.size(3), block.x);
    grid.z = src.size(1)*src.size(0);

    assert(grid.y < 65536);
    assert(grid.z < 65536);

    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(src.device());
    torch::Tensor dst = torch::empty({src.size(0), src.size(3), src.size(1), src.size(2)}, options);

    const int ptxv = getPtxver();

    // to be further specialized for additional archs, if necessary
    if (ptxv < 100) {
        block.y = TRANSP_WARPS_X_TILE_GENERIC;
        permute_to0312_k<WARP_SIZE, TRANSP_WARPS_X_TILE_GENERIC>
                        <<<grid, block, 0, stream>>>(src.size(3),
                                                     src.size(1),
                                                     src.size(2),
                                                     src.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
                                                     dst.packed_accessor32<float, 4, torch::RestrictPtrTraits>());
        CHECK_ERROR("permute_to0312_k_tile_generic");
    } else {
        block.y = TRANSP_WARPS_X_TILE_SM100;
        permute_to0312_k<WARP_SIZE, TRANSP_WARPS_X_TILE_SM100>
                        <<<grid, block, 0, stream>>>(src.size(3),
                                                     src.size(1),
                                                     src.size(2),
                                                     src.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
                                                     dst.packed_accessor32<float, 4, torch::RestrictPtrTraits>());
        CHECK_ERROR("permute_to0312_k_tile_sm100");
    }

    return dst;
}
// END - tensor permutation kernels and functions

// BEGIN - general host-side functions
unsigned int next_pow2(unsigned int x) { 

    x -= 1;

    #pragma unroll
    for(int i = 1; i <= sizeof(x)*8 / 2; i *= 2) {
        x |= x >> i;    
    }
    return x+1;
}
// END - general host-side functions
